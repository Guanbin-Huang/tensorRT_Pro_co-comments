#include "hip/hip_runtime.h"

#include "simple_yolo.hpp"
#include <NvInfer.h>
#include <NvOnnxParser.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <fstream>
#include <memory>
#include <string>
#include <future>
#include <condition_variable>
#include <mutex>
#include <thread>
#include <queue>

#if defined(_WIN32)
#	include <Windows.h>
#   include <wingdi.h>
#	include <Shlwapi.h>
#	pragma comment(lib, "shlwapi.lib")
#	undef min
#	undef max
#else
#	include <dirent.h>
#	include <sys/types.h>
#	include <sys/stat.h>
#	include <unistd.h>
#   include <stdarg.h>
#endif

namespace SimpleYolo{

    using namespace nvinfer1;
    using namespace std;
    using namespace cv;

    #define CURRENT_DEVICE_ID   -1
    #define GPU_BLOCK_THREADS  512
    #define KernelPositionBlock											\
        int position = (blockDim.x * blockIdx.x + threadIdx.x);		    \
        if (position >= (edge)) return;

    #define checkCudaRuntime(call) check_runtime(call, #call, __LINE__, __FILE__)
    static bool check_runtime(hipError_t e, const char* call, int line, const char *file);

    #define checkCudaKernel(...)                                                                         \
        __VA_ARGS__;                                                                                     \
        do{hipError_t cudaStatus = hipPeekAtLastError();                                               \
        if (cudaStatus != hipSuccess){                                                                  \
            INFOE("launch failed: %s", hipGetErrorString(cudaStatus));                                  \
        }} while(0);

    #define Assert(op)					 \
        do{                              \
            bool cond = !(!(op));        \
            if(!cond){                   \
                INFOF("Assert failed, " #op);  \
            }                                  \
        }while(false)

    /* 修改这个level来实现修改日志输出级别 */
    #define CURRENT_LOG_LEVEL       LogLevel::Info
    #define INFOD(...)			__log_func(__FILE__, __LINE__, LogLevel::Debug, __VA_ARGS__)
    #define INFOV(...)			__log_func(__FILE__, __LINE__, LogLevel::Verbose, __VA_ARGS__)
    #define INFO(...)			__log_func(__FILE__, __LINE__, LogLevel::Info, __VA_ARGS__)
    #define INFOW(...)			__log_func(__FILE__, __LINE__, LogLevel::Warning, __VA_ARGS__)
    #define INFOE(...)			__log_func(__FILE__, __LINE__, LogLevel::Error, __VA_ARGS__)
    #define INFOF(...)			__log_func(__FILE__, __LINE__, LogLevel::Fatal, __VA_ARGS__)

    enum class NormType : int{
        None      = 0,
        MeanStd   = 1,
        AlphaBeta = 2
    };

    enum class ChannelType : int{
        None          = 0,
        SwapRB        = 1
    };

    /* 归一化操作，可以支持均值标准差，alpha beta，和swap RB */
    struct Norm{
        float mean[3];
        float std[3];
        float alpha, beta;
        NormType type = NormType::None;
        ChannelType channel_type = ChannelType::None;

        // out = (x * alpha - mean) / std
        static Norm mean_std(const float mean[3], const float std[3], float alpha = 1/255.0f, ChannelType channel_type=ChannelType::None);

        // out = x * alpha + beta
        static Norm alpha_beta(float alpha, float beta = 0, ChannelType channel_type=ChannelType::None);

        // None
        static Norm None();
    };

    Norm Norm::mean_std(const float mean[3], const float std[3], float alpha, ChannelType channel_type){

        Norm out;
        out.type  = NormType::MeanStd;
        out.alpha = alpha;
        out.channel_type = channel_type;
        memcpy(out.mean, mean, sizeof(out.mean));
        memcpy(out.std,  std,  sizeof(out.std));
        return out;
    }

    Norm Norm::alpha_beta(float alpha, float beta, ChannelType channel_type){

        Norm out;
        out.type = NormType::AlphaBeta;
        out.alpha = alpha;
        out.beta = beta;
        out.channel_type = channel_type;
        return out;
    }

    Norm Norm::None(){
        return Norm();
    }

    /* 构造时设置当前gpuid，析构时修改为原来的gpuid */
    class AutoDevice{
    public:
        AutoDevice(int device_id = 0){
            hipGetDevice(&old_);
            checkCudaRuntime(hipSetDevice(device_id));
        }

        virtual ~AutoDevice(){
            checkCudaRuntime(hipSetDevice(old_));
        }
    
    private:
        int old_ = -1;
    };
    
    enum class LogLevel : int{
        Debug   = 5,
        Verbose = 4,
        Info    = 3,
        Warning = 2,
        Error   = 1,
        Fatal   = 0
    };

    static void __log_func(const char* file, int line, LogLevel level, const char* fmt, ...);
    inline int upbound(int n, int align = 32){return (n + align - 1) / align * align;}

    static bool check_runtime(hipError_t e, const char* call, int line, const char *file){
        if (e != hipSuccess) {
            INFOE("CUDA Runtime error %s # %s, code = %s [ %d ] in file %s:%d", call, hipGetErrorString(e), hipGetErrorName(e), e, file, line);
            return false;
        }
        return true;
    }

    #define TRT_STR(v)  #v
    #define TRT_VERSION_STRING(major, minor, patch, build)   TRT_STR(major) "." TRT_STR(minor) "." TRT_STR(patch) "." TRT_STR(build)
    const char* trt_version(){
        return TRT_VERSION_STRING(NV_TENSORRT_MAJOR, NV_TENSORRT_MINOR, NV_TENSORRT_PATCH, NV_TENSORRT_BUILD);
    }

    static bool check_device_id(int device_id){
        int device_count = -1;
        checkCudaRuntime(hipGetDeviceCount(&device_count));
        if(device_id < 0 || device_id >= device_count){
            INFOE("Invalid device id: %d, count = %d", device_id, device_count);
            return false;
        }
        return true;
    }

    static bool exists(const string& path){

    #ifdef _WIN32
        return ::PathFileExistsA(path.c_str());
    #else
        return access(path.c_str(), R_OK) == 0;
    #endif
    }

    static const char* level_string(LogLevel level){
        switch (level){
            case LogLevel::Debug: return "debug";
            case LogLevel::Verbose: return "verbo";
            case LogLevel::Info: return "info";
            case LogLevel::Warning: return "warn";
            case LogLevel::Error: return "error";
            case LogLevel::Fatal: return "fatal";
            default: return "unknow";
        }
    }

    template<typename _T>
    static string join_dims(const vector<_T>& dims){
        stringstream output;
        char buf[64];
        const char* fmts[] = {"%d", " x %d"};
        for(int i = 0; i < dims.size(); ++i){
            snprintf(buf, sizeof(buf), fmts[i != 0], dims[i]);
            output << buf;
        }
        return output.str();
    }

    static bool save_file(const string& file, const void* data, size_t length){

        FILE* f = fopen(file.c_str(), "wb");
        if (!f) return false;

        if (data && length > 0){
            if (fwrite(data, 1, length, f) != length){
                fclose(f);
                return false;
            }
        }
        fclose(f);
        return true;
    }

    static bool save_file(const string& file, const vector<uint8_t>& data){
        return save_file(file, data.data(), data.size());
    }

    static string file_name(const string& path, bool include_suffix){

        if (path.empty()) return "";

        int p = path.rfind('/');

#ifdef U_OS_WINDOWS
        int e = path.rfind('\\');
        p = std::max(p, e);
#endif
        p += 1;

        //include suffix
        if (include_suffix)
            return path.substr(p);

        int u = path.rfind('.');
        if (u == -1)
            return path.substr(p);

        if (u <= p) u = path.size();
        return path.substr(p, u - p);
    }

    vector<string> glob_image_files(const string& directory){

        /* 检索目录下的所有图像："*.jpg;*.png;*.bmp;*.jpeg;*.tiff" */
        vector<string> files, output;
        set<string> pattern_set{"jpg", "png", "bmp", "jpeg", "tiff"};

        if(directory.empty()){
            INFOE("Glob images from folder failed, folder is empty");
            return output;
        }

        try{
			vector<cv::String> files_;
			files_.reserve(10000);
            cv::glob(directory + "/*", files_, true);
			files.insert(files.end(), files_.begin(), files_.end());
        }catch(...){
            INFOE("Glob %s failed", directory.c_str());
            return output;
        }

        for(int i = 0; i < files.size(); ++i){
            auto& file = files[i];
            int p = file.rfind(".");
            if(p == -1) continue;

            auto suffix = file.substr(p+1);
            std::transform(suffix.begin(), suffix.end(), suffix.begin(), [](char c){
                if(c >= 'A' && c <= 'Z')
                    c -= 'A' + 'a';
                return c;
            });
            if(pattern_set.find(suffix) != pattern_set.end())
                output.push_back(file);
        }
        return output;
    }

    static void __log_func(const char* file, int line, LogLevel level, const char* fmt, ...){

        if(level > CURRENT_LOG_LEVEL)
            return;

        va_list vl;
        va_start(vl, fmt);
        
        char buffer[2048];
        string filename = file_name(file, true);
        int n = snprintf(buffer, sizeof(buffer), "[%s][%s:%d]:", level_string(level), filename.c_str(), line);
        vsnprintf(buffer + n, sizeof(buffer) - n, fmt, vl);

        fprintf(stdout, "%s\n", buffer);
        if (level == LogLevel::Fatal) {
            fflush(stdout);
            abort();
        }
    }

    static dim3 grid_dims(int numJobs) {
        int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
        return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
    }

    static dim3 block_dims(int numJobs) {
        return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
    }

    static int get_device(int device_id){
        if(device_id != CURRENT_DEVICE_ID){
            check_device_id(device_id);
            return device_id;
        }

        checkCudaRuntime(hipGetDevice(&device_id));
        return device_id;
    }

    void set_device(int device_id) {
        if (device_id == -1)
            return;

        checkCudaRuntime(hipSetDevice(device_id));
    }

    /////////////////////////////CUDA kernels////////////////////////////////////////////////

    const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
    static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
        *ox = matrix[0] * x + matrix[1] * y + matrix[2];
        *oy = matrix[3] * x + matrix[4] * y + matrix[5];
    }

    static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects){  

        int position = blockDim.x * blockIdx.x + threadIdx.x;
        if (position >= num_bboxes) return;

        float* pitem     = predict + (5 + num_classes) * position;
        float objectness = pitem[4];
        if(objectness < confidence_threshold)
            return;

        float* class_confidence = pitem + 5;
        float confidence        = *class_confidence++;
        int label               = 0;
        for(int i = 1; i < num_classes; ++i, ++class_confidence){
            if(*class_confidence > confidence){
                confidence = *class_confidence;
                label      = i;
            }
        }

        confidence *= objectness;
        if(confidence < confidence_threshold)
            return;

        int index = atomicAdd(parray, 1);
        if(index >= max_objects)
            return;

        float cx         = *pitem++;
        float cy         = *pitem++;
        float width      = *pitem++;
        float height     = *pitem++;
        float left   = cx - width * 0.5f;
        float top    = cy - height * 0.5f;
        float right  = cx + width * 0.5f;
        float bottom = cy + height * 0.5f;
        affine_project(invert_affine_matrix, left,  top,    &left,  &top);
        affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

        float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
        *pout_item++ = left;
        *pout_item++ = top;
        *pout_item++ = right;
        *pout_item++ = bottom;
        *pout_item++ = confidence;
        *pout_item++ = label;
        *pout_item++ = 1; // 1 = keep, 0 = ignore
    }

    static __device__ float box_iou(
        float aleft, float atop, float aright, float abottom, 
        float bleft, float btop, float bright, float bbottom
    ){

        float cleft 	= max(aleft, bleft);
        float ctop 		= max(atop, btop);
        float cright 	= min(aright, bright);
        float cbottom 	= min(abottom, bbottom);
        
        float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
        if(c_area == 0.0f)
            return 0.0f;
        
        float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
        float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
        return c_area / (a_area + b_area - c_area);
    }
    
    static __global__ void fast_nms_kernel(float* bboxes, int max_objects, float threshold){

        int position = (blockDim.x * blockIdx.x + threadIdx.x);
        int count = min((int)*bboxes, max_objects);
        if (position >= count) 
            return;
        
        // left, top, right, bottom, confidence, class, keepflag
        float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
        for(int i = 0; i < count; ++i){
            float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
            if(i == position || pcurrent[5] != pitem[5]) continue;

            if(pitem[4] >= pcurrent[4]){
                if(pitem[4] == pcurrent[4] && i < position)
                    continue;

                float iou = box_iou(
                    pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                    pitem[0],    pitem[1],    pitem[2],    pitem[3]
                );

                if(iou > threshold){
                    pcurrent[6] = 0;  // 1=keep, 0=ignore
                    return;
                }
            }
        }
    } 

    static void decode_kernel_invoker(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float nms_threshold, float* invert_affine_matrix, float* parray, int max_objects, hipStream_t stream){
        
        auto grid = grid_dims(num_bboxes);
        auto block = block_dims(num_bboxes);

        /* 如果核函数有波浪线，没关系，他是正常的，你只是看不顺眼罢了 */
        checkCudaKernel(decode_kernel<<<grid, block, 0, stream>>>(predict, num_bboxes, num_classes, confidence_threshold, invert_affine_matrix, parray, max_objects));

        grid = grid_dims(max_objects);
        block = block_dims(max_objects);
        checkCudaKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold));
    }

    static __global__ void warp_affine_bilinear_and_normalize_plane_kernel(uint8_t* src, int src_line_size, int src_width, int src_height, float* dst, int dst_width, int dst_height, 
        uint8_t const_value_st, float* warp_affine_matrix_2_3, Norm norm, int edge){

        int position = blockDim.x * blockIdx.x + threadIdx.x;
        if (position >= edge) return;

        float m_x1 = warp_affine_matrix_2_3[0];
        float m_y1 = warp_affine_matrix_2_3[1];
        float m_z1 = warp_affine_matrix_2_3[2];
        float m_x2 = warp_affine_matrix_2_3[3];
        float m_y2 = warp_affine_matrix_2_3[4];
        float m_z2 = warp_affine_matrix_2_3[5];

        int dx      = position % dst_width;
        int dy      = position / dst_width;
        float src_x = m_x1 * dx + m_y1 * dy + m_z1;
        float src_y = m_x2 * dx + m_y2 * dy + m_z2;
        float c0, c1, c2;

        if(src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height){
            // out of range
            c0 = const_value_st;
            c1 = const_value_st;
            c2 = const_value_st;
        }else{
            int y_low = floorf(src_y);
            int x_low = floorf(src_x);
            int y_high = y_low + 1;
            int x_high = x_low + 1;

            uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
            float ly    = src_y - y_low;
            float lx    = src_x - x_low;
            float hy    = 1 - ly;
            float hx    = 1 - lx;
            float w1    = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
            uint8_t* v1 = const_value;
            uint8_t* v2 = const_value;
            uint8_t* v3 = const_value;
            uint8_t* v4 = const_value;
            if(y_low >= 0){
                if (x_low >= 0)
                    v1 = src + y_low * src_line_size + x_low * 3;

                if (x_high < src_width)
                    v2 = src + y_low * src_line_size + x_high * 3;
            }
            
            if(y_high < src_height){
                if (x_low >= 0)
                    v3 = src + y_high * src_line_size + x_low * 3;

                if (x_high < src_width)
                    v4 = src + y_high * src_line_size + x_high * 3;
            }

            // same to opencv
            c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
            c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
            c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
        }

        if(norm.channel_type == ChannelType::SwapRB){
            float t = c2;
            c2 = c0;  c0 = t;
        }

        if(norm.type == NormType::MeanStd){
            c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
            c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
            c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
        }else if(norm.type == NormType::AlphaBeta){
            c0 = c0 * norm.alpha + norm.beta;
            c1 = c1 * norm.alpha + norm.beta;
            c2 = c2 * norm.alpha + norm.beta;
        }

        int area = dst_width * dst_height;
        float* pdst_c0 = dst + dy * dst_width + dx;
        float* pdst_c1 = pdst_c0 + area;
        float* pdst_c2 = pdst_c1 + area;
        *pdst_c0 = c0;
        *pdst_c1 = c1;
        *pdst_c2 = c2;
    }

    static void warp_affine_bilinear_and_normalize_plane(
        uint8_t* src, int src_line_size, int src_width, int src_height, float* dst, int dst_width, int dst_height,
        float* matrix_2_3, uint8_t const_value, const Norm& norm,
        hipStream_t stream) {
        
        int jobs   = dst_width * dst_height;
        auto grid  = grid_dims(jobs);
        auto block = block_dims(jobs);
        
        checkCudaKernel(warp_affine_bilinear_and_normalize_plane_kernel << <grid, block, 0, stream >> > (
            src, src_line_size,
            src_width, src_height, dst,
            dst_width, dst_height, const_value, matrix_2_3, norm, jobs
        ));
    }


    //////////////////////////////class MixMemory/////////////////////////////////////////////////
    /* gpu/cpu内存管理
        自动对gpu和cpu内存进行分配和释放
        这里的cpu使用的是pinned memory，当对gpu做内存复制时，性能比较好
        因为是hipHostMalloc分配的，因此他与cuda context有关联
    */
    class MixMemory {
    public:
        MixMemory(int device_id = CURRENT_DEVICE_ID);
        MixMemory(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size);
        virtual ~MixMemory();
        void* gpu(size_t size);
        void* cpu(size_t size);
        void release_gpu();
        void release_cpu();
        void release_all();

        inline bool owner_gpu() const{return owner_gpu_;}
        inline bool owner_cpu() const{return owner_cpu_;}

        inline size_t cpu_size() const{return cpu_size_;}
        inline size_t gpu_size() const{return gpu_size_;}
        inline int device_id() const{return device_id_;}

        inline void* gpu() const { return gpu_; }

        // Pinned Memory
        inline void* cpu() const { return cpu_; }

        void reference_data(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size);

    private:
        void* cpu_ = nullptr;
        size_t cpu_size_ = 0;
        bool owner_cpu_ = true;
        int device_id_ = 0;

        void* gpu_ = nullptr;
        size_t gpu_size_ = 0;
        bool owner_gpu_ = true;
    };

    MixMemory::MixMemory(int device_id){
        device_id_ = get_device(device_id);
    }

    MixMemory::MixMemory(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size){
        reference_data(cpu, cpu_size, gpu, gpu_size);		
    }

    void MixMemory::reference_data(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size){
        release_all();
        
        if(cpu == nullptr || cpu_size == 0){
            cpu = nullptr;
            cpu_size = 0;
        }

        if(gpu == nullptr || gpu_size == 0){
            gpu = nullptr;
            gpu_size = 0;
        }

        this->cpu_ = cpu;
        this->cpu_size_ = cpu_size;
        this->gpu_ = gpu;
        this->gpu_size_ = gpu_size;

        this->owner_cpu_ = !(cpu && cpu_size > 0);
        this->owner_gpu_ = !(gpu && gpu_size > 0);
        checkCudaRuntime(hipGetDevice(&device_id_));
    }

    MixMemory::~MixMemory() {
        release_all();
    }

    void* MixMemory::gpu(size_t size) {

        if (gpu_size_ < size) {
            release_gpu();

            gpu_size_ = size;
            AutoDevice auto_device_exchange(device_id_);
            checkCudaRuntime(hipMalloc(&gpu_, size));
            checkCudaRuntime(hipMemset(gpu_, 0, size));
        }
        return gpu_;
    }

    void* MixMemory::cpu(size_t size) {

        if (cpu_size_ < size) {
            release_cpu();

            cpu_size_ = size;
            AutoDevice auto_device_exchange(device_id_);
            checkCudaRuntime(hipHostMalloc(&cpu_, size));
            Assert(cpu_ != nullptr);
            memset(cpu_, 0, size);
        }
        return cpu_;
    }

    void MixMemory::release_cpu() {
        if (cpu_) {
            if(owner_cpu_){
                AutoDevice auto_device_exchange(device_id_);
                checkCudaRuntime(hipHostFree(cpu_));
            }
            cpu_ = nullptr;
        }
        cpu_size_ = 0;
    }

    void MixMemory::release_gpu() {
        if (gpu_) {
            if(owner_gpu_){
                AutoDevice auto_device_exchange(device_id_);
                checkCudaRuntime(hipFree(gpu_));
            }
            gpu_ = nullptr;
        }
        gpu_size_ = 0;
    }

    void MixMemory::release_all() {
        release_cpu();
        release_gpu();
    }

    /////////////////////////////////class Tensor////////////////////////////////////////////////
    /* Tensor类，实现张量的管理
        由于NN多用张量，必须有个类进行管理才方便，实现内存自动分配，计算索引等等
        如果要调试，可以执行save_to_file，储存为文件后，在python中加载并查看
    */
    enum class DataHead : int{
        Init   = 0,
        Device = 1,
        Host   = 2
    };

    class Tensor {
    public:
        Tensor(const Tensor& other) = delete;
        Tensor& operator = (const Tensor& other) = delete;

        explicit Tensor(std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        explicit Tensor(int n, int c, int h, int w, std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        explicit Tensor(int ndims, const int* dims, std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        explicit Tensor(const std::vector<int>& dims, std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        virtual ~Tensor();

        int numel() const;
        inline int ndims() const{return shape_.size();}
        inline int size(int index)  const{return shape_[index];}
        inline int shape(int index) const{return shape_[index];}

        inline int batch()   const{return shape_[0];}
        inline int channel() const{return shape_[1];}
        inline int height()  const{return shape_[2];}
        inline int width()   const{return shape_[3];}

        inline const std::vector<int>& dims() const { return shape_; }
        inline int bytes()                    const { return bytes_; }
        inline int bytes(int start_axis)      const { return count(start_axis) * element_size(); }
        inline int element_size()             const { return sizeof(float); }
        inline DataHead head()                const { return head_; }

        std::shared_ptr<Tensor> clone() const;
        Tensor& release();
        Tensor& set_to(float value);
        bool empty() const;

        template<typename ... _Args>
        int offset(int index, _Args ... index_args) const{
            const int index_array[] = {index, index_args...};
            return offset_array(sizeof...(index_args) + 1, index_array);
        }

        int offset_array(const std::vector<int>& index) const;
        int offset_array(size_t size, const int* index_array) const;

        template<typename ... _Args>
        Tensor& resize(int dim_size, _Args ... dim_size_args){
            const int dim_size_array[] = {dim_size, dim_size_args...};
            return resize(sizeof...(dim_size_args) + 1, dim_size_array);
        }

        Tensor& resize(int ndims, const int* dims);
        Tensor& resize(const std::vector<int>& dims);
        Tensor& resize_single_dim(int idim, int size);
        int  count(int start_axis = 0) const;
        int device() const{return device_id_;}

        Tensor& to_gpu(bool copy=true);
        Tensor& to_cpu(bool copy=true);

        inline void* cpu() const { ((Tensor*)this)->to_cpu(); return data_->cpu(); }
        inline void* gpu() const { ((Tensor*)this)->to_gpu(); return data_->gpu(); }
        
        template<typename DType> inline const DType* cpu() const { return (DType*)cpu(); }
        template<typename DType> inline DType* cpu()             { return (DType*)cpu(); }

        template<typename DType, typename ... _Args> 
        inline DType* cpu(int i, _Args&& ... args) { return cpu<DType>() + offset(i, args...); }


        template<typename DType> inline const DType* gpu() const { return (DType*)gpu(); }
        template<typename DType> inline DType* gpu()             { return (DType*)gpu(); }

        template<typename DType, typename ... _Args> 
        inline DType* gpu(int i, _Args&& ... args) { return gpu<DType>() + offset(i, args...); }

        template<typename DType, typename ... _Args> 
        inline DType& at(int i, _Args&& ... args) { return *(cpu<DType>() + offset(i, args...)); }
        
        std::shared_ptr<MixMemory> get_data()             const {return data_;}
        std::shared_ptr<MixMemory> get_workspace()        const {return workspace_;}
        Tensor& set_workspace(std::shared_ptr<MixMemory> workspace) {workspace_ = workspace; return *this;}

        hipStream_t get_stream() const{return stream_;}
        Tensor& set_stream(hipStream_t stream){stream_ = stream; return *this;}

        Tensor& set_mat     (int n, const cv::Mat& image);
        Tensor& set_norm_mat(int n, const cv::Mat& image, float mean[3], float std[3]);
        cv::Mat at_mat(int n = 0, int c = 0) { return cv::Mat(height(), width(), CV_32F, cpu<float>(n, c)); }

        Tensor& synchronize();
        const char* shape_string() const{return shape_string_;}
        const char* descriptor() const;

        Tensor& copy_from_gpu(size_t offset, const void* src, size_t num_element, int device_id = CURRENT_DEVICE_ID);

        /**
        
        # 以下代码是python中加载Tensor
        import numpy as np

        def load_tensor(file):
            
            with open(file, "rb") as f:
                binary_data = f.read()

            magic_number, ndims, dtype = np.frombuffer(binary_data, np.uint32, count=3, offset=0)
            assert magic_number == 0xFCCFE2E2, f"{file} not a tensor file."
            
            dims = np.frombuffer(binary_data, np.uint32, count=ndims, offset=3 * 4)

            if dtype == 0:
                np_dtype = np.float32
            elif dtype == 1:
                np_dtype = np.float16
            else:
                assert False, f"Unsupport dtype = {dtype}, can not convert to numpy dtype"
                
            return np.frombuffer(binary_data, np_dtype, offset=(ndims + 3) * 4).reshape(*dims)

         **/
        bool save_to_file(const std::string& file) const;

    private:
        Tensor& compute_shape_string();
        Tensor& adajust_memory_by_update_dims_or_type();
        void setup_data(std::shared_ptr<MixMemory> data);

    private:
        std::vector<int> shape_;
        size_t bytes_    = 0;
        DataHead head_   = DataHead::Init;
        hipStream_t stream_ = nullptr;
        int device_id_   = 0;
        char shape_string_[100];
        char descriptor_string_[100];
        std::shared_ptr<MixMemory> data_;
        std::shared_ptr<MixMemory> workspace_;
    };

    Tensor::Tensor(int n, int c, int h, int w, shared_ptr<MixMemory> data, int device_id) {
        this->device_id_ = get_device(device_id);
        descriptor_string_[0] = 0;
        setup_data(data);
        resize(n, c, h, w);
    }

    Tensor::Tensor(const std::vector<int>& dims, shared_ptr<MixMemory> data, int device_id){
        this->device_id_ = get_device(device_id);
        descriptor_string_[0] = 0;
        setup_data(data);
        resize(dims);
    }

    Tensor::Tensor(int ndims, const int* dims, shared_ptr<MixMemory> data, int device_id) {
        this->device_id_ = get_device(device_id);
        descriptor_string_[0] = 0;
        setup_data(data);
        resize(ndims, dims);
    }

    Tensor::Tensor(shared_ptr<MixMemory> data, int device_id){
        shape_string_[0] = 0;
        descriptor_string_[0] = 0;
        this->device_id_ = get_device(device_id);
        setup_data(data);
    }

    Tensor::~Tensor() {
        release();
    }

    const char* Tensor::descriptor() const{
        
        char* descriptor_ptr = (char*)descriptor_string_;
        int device_id = device();
        snprintf(descriptor_ptr, sizeof(descriptor_string_), 
            "Tensor:%p, %s, CUDA:%d", 
            data_.get(),
            shape_string_, 
            device_id
        );
        return descriptor_ptr;
    }

    Tensor& Tensor::compute_shape_string(){

        // clean string
        shape_string_[0] = 0;

        char* buffer = shape_string_;
        size_t buffer_size = sizeof(shape_string_);
        for(int i = 0; i < shape_.size(); ++i){

            int size = 0;
            if(i < shape_.size() - 1)
                size = snprintf(buffer, buffer_size, "%d x ", shape_[i]);
            else
                size = snprintf(buffer, buffer_size, "%d", shape_[i]);

            buffer += size;
            buffer_size -= size;
        }
        return *this;
    }

    void Tensor::setup_data(shared_ptr<MixMemory> data){
        
        data_ = data;
        if(data_ == nullptr){
            data_ = make_shared<MixMemory>(device_id_);
        }else{
            device_id_ = data_->device_id();
        }

        head_ = DataHead::Init;
        if(data_->cpu()){
            head_ = DataHead::Host;
        }

        if(data_->gpu()){
            head_ = DataHead::Device;
        }
    }

    Tensor& Tensor::copy_from_gpu(size_t offset, const void* src, size_t num_element, int device_id){

        if(head_ == DataHead::Init)
            to_gpu(false);

        size_t offset_location = offset * element_size();
        if(offset_location >= bytes_){
            INFOE("Offset location[%lld] >= bytes_[%lld], out of range", offset_location, bytes_);
            return *this;
        }

        size_t copyed_bytes = num_element * element_size();
        size_t remain_bytes = bytes_ - offset_location;
        if(copyed_bytes > remain_bytes){
            INFOE("Copyed bytes[%lld] > remain bytes[%lld], out of range", copyed_bytes, remain_bytes);
            return *this;
        }
        
        if(head_ == DataHead::Device){
            int current_device_id = get_device(device_id);
            int gpu_device_id = device();
            if(current_device_id != gpu_device_id){
                checkCudaRuntime(hipMemcpyPeerAsync(gpu<unsigned char>() + offset_location, gpu_device_id, src, current_device_id, copyed_bytes, stream_));
                //checkCudaRuntime(hipMemcpyAsync(gpu<unsigned char>() + offset_location, src, copyed_bytes, hipMemcpyDeviceToDevice, stream_));
            }
            else{
                checkCudaRuntime(hipMemcpyAsync(gpu<unsigned char>() + offset_location, src, copyed_bytes, hipMemcpyDeviceToDevice, stream_));
            }
        }else if(head_ == DataHead::Host){
            AutoDevice auto_device_exchange(this->device());
            checkCudaRuntime(hipMemcpyAsync(cpu<unsigned char>() + offset_location, src, copyed_bytes, hipMemcpyDeviceToHost, stream_));
        }else{
            INFOE("Unsupport head type %d", head_);
        }
        return *this;
    }

    Tensor& Tensor::release() {
        data_->release_all();
        shape_.clear();
        bytes_ = 0;
        head_ = DataHead::Init;
        return *this;
    }

    bool Tensor::empty() const{
        return data_->cpu() == nullptr && data_->gpu() == nullptr;
    }

    int Tensor::count(int start_axis) const {

        if(start_axis >= 0 && start_axis < shape_.size()){
            int size = 1;
            for (int i = start_axis; i < shape_.size(); ++i) 
                size *= shape_[i];
            return size;
        }else{
            return 0;
        }
    }

    Tensor& Tensor::resize(const std::vector<int>& dims) {
        return resize(dims.size(), dims.data());
    }

    int Tensor::numel() const{
        int value = shape_.empty() ? 0 : 1;
        for(int i = 0; i < shape_.size(); ++i){
            value *= shape_[i];
        }
        return value;
    }

    Tensor& Tensor::resize_single_dim(int idim, int size){

        Assert(idim >= 0 && idim < shape_.size());

        auto new_shape = shape_;
        new_shape[idim] = size;
        return resize(new_shape);
    }

    Tensor& Tensor::resize(int ndims, const int* dims) {

        vector<int> setup_dims(ndims);
        for(int i = 0; i < ndims; ++i){
            int dim = dims[i];
            if(dim == -1){
                Assert(ndims == shape_.size());
                dim = shape_[i];
            }
            setup_dims[i] = dim;
        }
        this->shape_ = setup_dims;
        this->adajust_memory_by_update_dims_or_type();
        this->compute_shape_string();
        return *this;
    }

    Tensor& Tensor::adajust_memory_by_update_dims_or_type(){
        
        int needed_size = this->numel() * element_size();
        if(needed_size > this->bytes_){
            head_ = DataHead::Init;
        }
        this->bytes_ = needed_size;
        return *this;
    }

    Tensor& Tensor::synchronize(){ 
        AutoDevice auto_device_exchange(this->device());
        checkCudaRuntime(hipStreamSynchronize(stream_));
        return *this;
    }

    Tensor& Tensor::to_gpu(bool copy) {

        if (head_ == DataHead::Device)
            return *this;

        head_ = DataHead::Device;
        data_->gpu(bytes_);

        if (copy && data_->cpu() != nullptr) {
            AutoDevice auto_device_exchange(this->device());
            checkCudaRuntime(hipMemcpyAsync(data_->gpu(), data_->cpu(), bytes_, hipMemcpyHostToDevice, stream_));
        }
        return *this;
    }
    
    Tensor& Tensor::to_cpu(bool copy) {

        if (head_ == DataHead::Host)
            return *this;

        head_ = DataHead::Host;
        data_->cpu(bytes_);

        if (copy && data_->gpu() != nullptr) {
            AutoDevice auto_device_exchange(this->device());
            checkCudaRuntime(hipMemcpyAsync(data_->cpu(), data_->gpu(), bytes_, hipMemcpyDeviceToHost, stream_));
            checkCudaRuntime(hipStreamSynchronize(stream_));
        }
        return *this;
    }

    int Tensor::offset_array(size_t size, const int* index_array) const{

        Assert(size <= shape_.size());
        int value = 0;
        for(int i = 0; i < shape_.size(); ++i){

            if(i < size)
                value += index_array[i];

            if(i + 1 < shape_.size())
                value *= shape_[i+1];
        }
        return value;
    }

    int Tensor::offset_array(const std::vector<int>& index_array) const{
        return offset_array(index_array.size(), index_array.data());
    }

    bool Tensor::save_to_file(const std::string& file) const{

        if(empty()) return false;

        FILE* f = fopen(file.c_str(), "wb");
        if(f == nullptr) return false;

        int ndims = this->ndims();
        int dtype_ = 0;
        unsigned int head[3] = {0xFCCFE2E2, ndims, static_cast<unsigned int>(dtype_)};
        fwrite(head, 1, sizeof(head), f);
        fwrite(shape_.data(), 1, sizeof(shape_[0]) * shape_.size(), f);
        fwrite(cpu(), 1, bytes_, f);
        fclose(f);
        return true;
    }

    /////////////////////////////////class TRTInferImpl////////////////////////////////////////////////
    class Logger : public ILogger {
    public:
        virtual void log(Severity severity, const char* msg) noexcept override {

            if (severity == Severity::kINTERNAL_ERROR) {
                INFOE("NVInfer INTERNAL_ERROR: %s", msg);
                abort();
            }else if (severity == Severity::kERROR) {
                INFOE("NVInfer: %s", msg);
            }
            else  if (severity == Severity::kWARNING) {
                INFOW("NVInfer: %s", msg);
            }
            else  if (severity == Severity::kINFO) {
                INFOD("NVInfer: %s", msg);
            }
            else {
                INFOD("%s", msg);
            }
        }
    };
    static Logger gLogger;

    template<typename _T>
    static void destroy_nvidia_pointer(_T* ptr) {
        if (ptr) ptr->destroy();
    }

    class EngineContext {
    public:
        virtual ~EngineContext() { destroy(); }

        void set_stream(hipStream_t stream){

            if(owner_stream_){
                if (stream_) {hipStreamDestroy(stream_);}
                owner_stream_ = false;
            }
            stream_ = stream;
        }

        bool build_model(const void* pdata, size_t size) {
            destroy();

            if(pdata == nullptr || size == 0)
                return false;

            owner_stream_ = true;
            checkCudaRuntime(hipStreamCreate(&stream_));
            if(stream_ == nullptr)
                return false;

            runtime_ = shared_ptr<IRuntime>(createInferRuntime(gLogger), destroy_nvidia_pointer<IRuntime>);
            if (runtime_ == nullptr)
                return false;

            engine_ = shared_ptr<ICudaEngine>(runtime_->deserializeCudaEngine(pdata, size, nullptr), destroy_nvidia_pointer<ICudaEngine>);
            if (engine_ == nullptr)
                return false;

            //runtime_->setDLACore(0);
            context_ = shared_ptr<IExecutionContext>(engine_->createExecutionContext(), destroy_nvidia_pointer<IExecutionContext>);
            return context_ != nullptr;
        }

    private:
        void destroy() {
            context_.reset();
            engine_.reset();
            runtime_.reset();

            if(owner_stream_){
                if (stream_) {hipStreamDestroy(stream_);}
            }
            stream_ = nullptr;
        }

    public:
        hipStream_t stream_ = nullptr;
        bool owner_stream_ = false;
        shared_ptr<IExecutionContext> context_;
        shared_ptr<ICudaEngine> engine_;
        shared_ptr<IRuntime> runtime_ = nullptr;
    };

    class TRTInferImpl{
    public:
        virtual ~TRTInferImpl();
        bool load(const std::string& file);
        bool load_from_memory(const void* pdata, size_t size);
        void destroy();
        void forward(bool sync);
        int get_max_batch_size();
        hipStream_t get_stream();
        void set_stream(hipStream_t stream);
        void synchronize();
        size_t get_device_memory_size();
        std::shared_ptr<MixMemory> get_workspace();
        std::shared_ptr<Tensor> input(int index = 0);
        std::string get_input_name(int index = 0);
        std::shared_ptr<Tensor> output(int index = 0);
        std::string get_output_name(int index = 0);
        std::shared_ptr<Tensor> tensor(const std::string& name);
        bool is_output_name(const std::string& name);
        bool is_input_name(const std::string& name);
        void set_input (int index, std::shared_ptr<Tensor> tensor);
        void set_output(int index, std::shared_ptr<Tensor> tensor);
        std::shared_ptr<std::vector<uint8_t>> serial_engine();

        void print();

        int num_output();
        int num_input();
        int device();

    private:
        void build_engine_input_and_outputs_mapper();

    private:
        std::vector<std::shared_ptr<Tensor>> inputs_;
        std::vector<std::shared_ptr<Tensor>> outputs_;
        std::vector<int> inputs_map_to_ordered_index_;
        std::vector<int> outputs_map_to_ordered_index_;
        std::vector<std::string> inputs_name_;
        std::vector<std::string> outputs_name_;
        std::vector<std::shared_ptr<Tensor>> orderdBlobs_;
        std::map<std::string, int> blobsNameMapper_;
        std::shared_ptr<EngineContext> context_;
        std::vector<void*> bindingsPtr_;
        std::shared_ptr<MixMemory> workspace_;
        int device_ = 0;
    };

    ////////////////////////////////////////////////////////////////////////////////////
    TRTInferImpl::~TRTInferImpl(){
        destroy();
    }

    void TRTInferImpl::destroy() {

        int old_device = 0;
        checkCudaRuntime(hipGetDevice(&old_device));
        checkCudaRuntime(hipSetDevice(device_));
        this->context_.reset();
        this->blobsNameMapper_.clear();
        this->outputs_.clear();
        this->inputs_.clear();
        this->inputs_name_.clear();
        this->outputs_name_.clear();
        checkCudaRuntime(hipSetDevice(old_device));
    }

    void TRTInferImpl::print(){
        if(!context_){
            INFOW("Infer print, nullptr.");
            return;
        }

        INFO("Infer %p detail", this);
        INFO("\tMax Batch Size: %d", this->get_max_batch_size());
        INFO("\tInputs: %d", inputs_.size());
        for(int i = 0; i < inputs_.size(); ++i){
            auto& tensor = inputs_[i];
            auto& name = inputs_name_[i];
            INFO("\t\t%d.%s : shape {%s}", i, name.c_str(), tensor->shape_string());
        }

        INFO("\tOutputs: %d", outputs_.size());
        for(int i = 0; i < outputs_.size(); ++i){
            auto& tensor = outputs_[i];
            auto& name = outputs_name_[i];
            INFO("\t\t%d.%s : shape {%s}", i, name.c_str(), tensor->shape_string());
        }
    }

    std::shared_ptr<std::vector<uint8_t>> TRTInferImpl::serial_engine() {
        auto memory = this->context_->engine_->serialize();
        auto output = make_shared<std::vector<uint8_t>>((uint8_t*)memory->data(), (uint8_t*)memory->data()+memory->size());
        memory->destroy();
        return output;
    }

    bool TRTInferImpl::load_from_memory(const void* pdata, size_t size) {

        if (pdata == nullptr || size == 0)
            return false;

        context_.reset(new EngineContext());

        //build model
        if (!context_->build_model(pdata, size)) {
            context_.reset();
            return false;
        }

        workspace_.reset(new MixMemory());
        hipGetDevice(&device_);
        build_engine_input_and_outputs_mapper();
        return true;
    }

    static std::vector<uint8_t> load_file(const string& file){

        ifstream in(file, ios::in | ios::binary);
        if (!in.is_open())
            return {};

        in.seekg(0, ios::end);
        size_t length = in.tellg();

        std::vector<uint8_t> data;
        if (length > 0){
            in.seekg(0, ios::beg);
            data.resize(length);

            in.read((char*)&data[0], length);
        }
        in.close();
        return data;
    }

    bool TRTInferImpl::load(const std::string& file) {

        auto data = load_file(file);
        if (data.empty())
            return false;

        context_.reset(new EngineContext());

        //build model
        if (!context_->build_model(data.data(), data.size())) {
            context_.reset();
            return false;
        }

        workspace_.reset(new MixMemory());
        hipGetDevice(&device_);
        build_engine_input_and_outputs_mapper();
        return true;
    }

    size_t TRTInferImpl::get_device_memory_size() {
        EngineContext* context = (EngineContext*)this->context_.get();
        return context->context_->getEngine().getDeviceMemorySize();
    }

    void TRTInferImpl::build_engine_input_and_outputs_mapper() {
        
        EngineContext* context = (EngineContext*)this->context_.get();
        int nbBindings = context->engine_->getNbBindings();
        int max_batchsize = context->engine_->getMaxBatchSize();

        inputs_.clear();
        inputs_name_.clear();
        outputs_.clear();
        outputs_name_.clear();
        orderdBlobs_.clear();
        bindingsPtr_.clear();
        blobsNameMapper_.clear();
        for (int i = 0; i < nbBindings; ++i) {

            auto dims = context->engine_->getBindingDimensions(i);
            auto type = context->engine_->getBindingDataType(i);
            const char* bindingName = context->engine_->getBindingName(i);
            dims.d[0] = max_batchsize;
            auto newTensor = make_shared<Tensor>(dims.nbDims, dims.d);
            newTensor->set_stream(this->context_->stream_);
            newTensor->set_workspace(this->workspace_);
            if (context->engine_->bindingIsInput(i)) {
                //if is input
                inputs_.push_back(newTensor);
                inputs_name_.push_back(bindingName);
                inputs_map_to_ordered_index_.push_back(orderdBlobs_.size());
            }
            else {
                //if is output
                outputs_.push_back(newTensor);
                outputs_name_.push_back(bindingName);
                outputs_map_to_ordered_index_.push_back(orderdBlobs_.size());
            }
            blobsNameMapper_[bindingName] = i;
            orderdBlobs_.push_back(newTensor);
        }
        bindingsPtr_.resize(orderdBlobs_.size());
    }

    void TRTInferImpl::set_stream(hipStream_t stream){
        this->context_->set_stream(stream);

        for(auto& t : orderdBlobs_)
            t->set_stream(stream);
    }

    hipStream_t TRTInferImpl::get_stream() {
        return this->context_->stream_;
    }

    int TRTInferImpl::device() {
        return device_;
    }

    void TRTInferImpl::synchronize() {
        checkCudaRuntime(hipStreamSynchronize(context_->stream_));
    }

    bool TRTInferImpl::is_output_name(const std::string& name){
        return std::find(outputs_name_.begin(), outputs_name_.end(), name) != outputs_name_.end();
    }

    bool TRTInferImpl::is_input_name(const std::string& name){
        return std::find(inputs_name_.begin(), inputs_name_.end(), name) != inputs_name_.end();
    }

    void TRTInferImpl::forward(bool sync) {

        EngineContext* context = (EngineContext*)context_.get();
        int inputBatchSize = inputs_[0]->size(0);
        for(int i = 0; i < context->engine_->getNbBindings(); ++i){
            auto dims = context->engine_->getBindingDimensions(i);
            auto type = context->engine_->getBindingDataType(i);
            dims.d[0] = inputBatchSize;
            if(context->engine_->bindingIsInput(i)){
                context->context_->setBindingDimensions(i, dims);
            }
        }

        for (int i = 0; i < outputs_.size(); ++i) {
            outputs_[i]->resize_single_dim(0, inputBatchSize);
            outputs_[i]->to_gpu(false);
        }

        for (int i = 0; i < orderdBlobs_.size(); ++i)
            bindingsPtr_[i] = orderdBlobs_[i]->gpu();

        void** bindingsptr = bindingsPtr_.data();
        //bool execute_result = context->context_->enqueue(inputBatchSize, bindingsptr, context->stream_, nullptr);
        bool execute_result = context->context_->enqueueV2(bindingsptr, context->stream_, nullptr);
        if(!execute_result){
            auto code = hipGetLastError();
            INFOF("execute fail, code %d[%s], message %s", code, hipGetErrorName(code), hipGetErrorString(code));
        }

        if (sync) {
            synchronize();
        }
    }

    std::shared_ptr<MixMemory> TRTInferImpl::get_workspace() {
        return workspace_;
    }

    int TRTInferImpl::num_input() {
        return this->inputs_.size();
    }

    int TRTInferImpl::num_output() {
        return this->outputs_.size();
    }

    void TRTInferImpl::set_input (int index, std::shared_ptr<Tensor> tensor){
        Assert(index >= 0 && index < inputs_.size());
        this->inputs_[index] = tensor;

        int order_index = inputs_map_to_ordered_index_[index];
        this->orderdBlobs_[order_index] = tensor;
    }

    void TRTInferImpl::set_output(int index, std::shared_ptr<Tensor> tensor){
        Assert(index >= 0 && index < outputs_.size());
        this->outputs_[index] = tensor;

        int order_index = outputs_map_to_ordered_index_[index];
        this->orderdBlobs_[order_index] = tensor;
    }

    std::shared_ptr<Tensor> TRTInferImpl::input(int index) {
        Assert(index >= 0 && index < inputs_name_.size());
        return this->inputs_[index];
    }

    std::string TRTInferImpl::get_input_name(int index){
        Assert(index >= 0 && index < inputs_name_.size());
        return inputs_name_[index];
    }

    std::shared_ptr<Tensor> TRTInferImpl::output(int index) {
        Assert(index >= 0 && index < outputs_.size());
        return outputs_[index];
    }

    std::string TRTInferImpl::get_output_name(int index){
        Assert(index >= 0 && index < outputs_name_.size());
        return outputs_name_[index];
    }

    int TRTInferImpl::get_max_batch_size() {
        Assert(this->context_ != nullptr);
        return this->context_->engine_->getMaxBatchSize();
    }

    std::shared_ptr<Tensor> TRTInferImpl::tensor(const std::string& name) {
        Assert(this->blobsNameMapper_.find(name) != this->blobsNameMapper_.end());
        return orderdBlobs_[blobsNameMapper_[name]];
    }

    std::shared_ptr<TRTInferImpl> load_infer(const string& file) {
        
        std::shared_ptr<TRTInferImpl> infer(new TRTInferImpl());
        if (!infer->load(file))
            infer.reset();
        return infer;
    }

    //////////////////////////////class MonopolyAllocator//////////////////////////////////////
    /* 独占分配器
       通过对tensor做独占管理，具有max_batch * 2个tensor，通过query获取一个
       当推理结束后，该tensor释放使用权，即可交给下一个图像使用，内存实现复用
    */
    template<class _ItemType>
    class MonopolyAllocator{
    public:
        class MonopolyData{
        public:
            std::shared_ptr<_ItemType>& data(){ return data_; }
            void release(){manager_->release_one(this);}

        private:
            MonopolyData(MonopolyAllocator* pmanager){manager_ = pmanager;}

        private:
            friend class MonopolyAllocator;
            MonopolyAllocator* manager_ = nullptr;
            std::shared_ptr<_ItemType> data_;
            bool available_ = true;
        };
        typedef std::shared_ptr<MonopolyData> MonopolyDataPointer;

        MonopolyAllocator(int size){
            capacity_ = size;
            num_available_ = size;
            datas_.resize(size);

            for(int i = 0; i < size; ++i)
                datas_[i] = std::shared_ptr<MonopolyData>(new MonopolyData(this));
        }

        virtual ~MonopolyAllocator(){
            run_ = false;
            cv_.notify_all();
            
            std::unique_lock<std::mutex> l(lock_);
            cv_exit_.wait(l, [&](){
                return num_wait_thread_ == 0;
            });
        }

        MonopolyDataPointer query(int timeout = 10000){

            std::unique_lock<std::mutex> l(lock_);
            if(!run_) return nullptr;
            
            if(num_available_ == 0){
                num_wait_thread_++;

                auto state = cv_.wait_for(l, std::chrono::milliseconds(timeout), [&](){
                    return num_available_ > 0 || !run_;
                });

                num_wait_thread_--;
                cv_exit_.notify_one();

                // timeout, no available, exit program
                if(!state || num_available_ == 0 || !run_)
                    return nullptr;
            }

            auto item = std::find_if(datas_.begin(), datas_.end(), [](MonopolyDataPointer& item){return item->available_;});
            if(item == datas_.end())
                return nullptr;
            
            (*item)->available_ = false;
            num_available_--;
            return *item;
        }

        int num_available(){
            return num_available_;
        }

        int capacity(){
            return capacity_;
        }

    private:
        void release_one(MonopolyData* prq){
            std::unique_lock<std::mutex> l(lock_);
            if(!prq->available_){
                prq->available_ = true;
                num_available_++;
                cv_.notify_one();
            }
        }

    private:
        std::mutex lock_;
        std::condition_variable cv_;
        std::condition_variable cv_exit_;
        std::vector<MonopolyDataPointer> datas_;
        int capacity_ = 0;
        volatile int num_available_ = 0;
        volatile int num_wait_thread_ = 0;
        volatile bool run_ = true;
    };


    /////////////////////////////////////////class ThreadSafedAsyncInfer/////////////////////////////////////////////
    /* 异步线程安全的推理器
       通过异步线程启动，使得调用方允许任意线程调用把图像做输入，并通过future来获取异步结果
    */
    template<class Input, class Output, class StartParam=std::tuple<std::string, int>, class JobAdditional=int>
    class ThreadSafedAsyncInfer{
    public:
        struct Job{
            Input input;
            Output output;
            JobAdditional additional;
            MonopolyAllocator<Tensor>::MonopolyDataPointer mono_tensor;
            std::shared_ptr<std::promise<Output>> pro;
        };

        virtual ~ThreadSafedAsyncInfer(){
            stop();
        }

        void stop(){
            run_ = false;
            cond_.notify_all();

            ////////////////////////////////////////// cleanup jobs
            {
                std::unique_lock<std::mutex> l(jobs_lock_);
                while(!jobs_.empty()){
                    auto& item = jobs_.front();
                    if(item.pro)
                        item.pro->set_value(Output());
                    jobs_.pop();
                }
            };

            if(worker_){
                worker_->join();
                worker_.reset();
            }
        }

        bool startup(const StartParam& param){
            run_ = true;

            std::promise<bool> pro;
            start_param_ = param;
            worker_      = std::make_shared<std::thread>(&ThreadSafedAsyncInfer::worker, this, std::ref(pro));
            return pro.get_future().get();
        }

        virtual std::shared_future<Output> commit(const Input& input){

            Job job;
            job.pro = std::make_shared<std::promise<Output>>();
            if(!preprocess(job, input)){
                job.pro->set_value(Output());
                return job.pro->get_future();
            }
            
            ///////////////////////////////////////////////////////////
            {
                std::unique_lock<std::mutex> l(jobs_lock_);
                jobs_.push(job);
            };
            cond_.notify_one();
            return job.pro->get_future();
        }

        virtual std::vector<std::shared_future<Output>> commits(const std::vector<Input>& inputs){

            int batch_size = std::min((int)inputs.size(), this->tensor_allocator_->capacity());
            std::vector<Job> jobs(inputs.size());
            std::vector<std::shared_future<Output>> results(inputs.size());

            int nepoch = (inputs.size() + batch_size - 1) / batch_size;
            for(int epoch = 0; epoch < nepoch; ++epoch){
                int begin = epoch * batch_size;
                int end   = std::min((int)inputs.size(), begin + batch_size);

                for(int i = begin; i < end; ++i){
                    Job& job = jobs[i];
                    job.pro = std::make_shared<std::promise<Output>>();
                    if(!preprocess(job, inputs[i])){
                        job.pro->set_value(Output());
                    }
                    results[i] = job.pro->get_future();
                }

                ///////////////////////////////////////////////////////////
                {
                    std::unique_lock<std::mutex> l(jobs_lock_);
                    for(int i = begin; i < end; ++i){
                        jobs_.emplace(std::move(jobs[i]));
                    };
                }
                cond_.notify_one();
            }
            return results;
        }

    protected:
        virtual void worker(std::promise<bool>& result) = 0;
        virtual bool preprocess(Job& job, const Input& input) = 0;
        
        virtual bool get_jobs_and_wait(std::vector<Job>& fetch_jobs, int max_size){

            std::unique_lock<std::mutex> l(jobs_lock_);
            cond_.wait(l, [&](){
                return !run_ || !jobs_.empty();
            });

            if(!run_) return false;
            
            fetch_jobs.clear();
            for(int i = 0; i < max_size && !jobs_.empty(); ++i){
                fetch_jobs.emplace_back(std::move(jobs_.front()));
                jobs_.pop();
            }
            return true;
        }

        virtual bool get_job_and_wait(Job& fetch_job){

            std::unique_lock<std::mutex> l(jobs_lock_);
            cond_.wait(l, [&](){
                return !run_ || !jobs_.empty();
            });

            if(!run_) return false;
            
            fetch_job = std::move(jobs_.front());
            jobs_.pop();
            return true;
        }

    protected:
        StartParam start_param_;
        std::atomic<bool> run_;
        std::mutex jobs_lock_;
        std::queue<Job> jobs_;
        std::shared_ptr<std::thread> worker_;
        std::condition_variable cond_;
        std::shared_ptr<MonopolyAllocator<Tensor>> tensor_allocator_;
    };


    ///////////////////////////////////class YoloTRTInferImpl//////////////////////////////////////
    /* Yolo的具体实现
        通过上述类的特性，实现预处理的计算重叠、异步垮线程调用，最终拼接为多个图为一个batch进行推理。最大化的利用
        显卡性能，实现高性能高可用好用的yolo推理
    */
    const char* type_name(Type type){
        switch(type){
        case Type::V5: return "YoloV5";
        case Type::X: return "YoloX";
        default: return "Unknow";
        }
    }

    struct AffineMatrix{
        float i2d[6];       // image to dst(network), 2x3 matrix
        float d2i[6];       // dst to image, 2x3 matrix

        void compute(const cv::Size& from, const cv::Size& to){
            float scale_x = to.width / (float)from.width;
            float scale_y = to.height / (float)from.height;
            float scale = std::min(scale_x, scale_y);
            i2d[0] = scale;  i2d[1] = 0;  i2d[2] = -scale * from.width  * 0.5  + to.width * 0.5 + scale * 0.5 - 0.5;
            i2d[3] = 0;  i2d[4] = scale;  i2d[5] = -scale * from.height * 0.5 + to.height * 0.5 + scale * 0.5 - 0.5;

            cv::Mat m2x3_i2d(2, 3, CV_32F, i2d);
            cv::Mat m2x3_d2i(2, 3, CV_32F, d2i);
            cv::invertAffineTransform(m2x3_i2d, m2x3_d2i);
        }

        cv::Mat i2d_mat(){
            return cv::Mat(2, 3, CV_32F, i2d);
        }
    };

    using ThreadSafedAsyncInferImpl = ThreadSafedAsyncInfer
    <
        cv::Mat,                    // input
        BoxArray,                   // output
        tuple<string, int>,         // start param
        AffineMatrix                // additional
    >;
    class YoloTRTInferImpl : public Infer, public ThreadSafedAsyncInferImpl{
    public:

        /** 要求在TRTInferImpl里面执行stop，而不是在基类执行stop **/
        virtual ~YoloTRTInferImpl(){
            stop();
        }

        virtual bool startup(const string& file, Type type, int gpuid, float confidence_threshold, float nms_threshold){

            if(type == Type::V5){
                normalize_ = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
            }else if(type == Type::X){
                //float mean[] = {0.485, 0.456, 0.406};
                //float std[]  = {0.229, 0.224, 0.225};
                //normalize_ = Norm::mean_std(mean, std, 1/255.0f, ChannelType::Invert);
                normalize_ = Norm::None();
            }else{
                INFOE("Unsupport type %d", type);
            }
            
            confidence_threshold_ = confidence_threshold;
            nms_threshold_        = nms_threshold;
            return ThreadSafedAsyncInferImpl::startup(make_tuple(file, gpuid));
        }

        virtual void worker(promise<bool>& result) override{

            string file = get<0>(start_param_);
            int gpuid   = get<1>(start_param_);

            set_device(gpuid);
            auto engine = load_infer(file);
            if(engine == nullptr){
                INFOE("Engine %s load failed", file.c_str());
                result.set_value(false);
                return;
            }

            engine->print();

            const int MAX_IMAGE_BBOX  = 1024;
            const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
            Tensor affin_matrix_device;
            Tensor output_array_device;
            int max_batch_size = engine->get_max_batch_size();
            auto input         = engine->tensor("images");
            auto output        = engine->tensor("output");
            int num_classes    = output->size(2) - 5;

            input_width_       = input->size(3);
            input_height_      = input->size(2);
            tensor_allocator_  = make_shared<MonopolyAllocator<Tensor>>(max_batch_size * 2);
            stream_            = engine->get_stream();
            gpu_               = gpuid;
            result.set_value(true);

            input->resize_single_dim(0, max_batch_size).to_gpu();
            affin_matrix_device.set_stream(stream_);

            // 这里8个值的目的是保证 8 * sizeof(float) % 32 == 0
            affin_matrix_device.resize(max_batch_size, 8).to_gpu();

            // 这里的 1 + MAX_IMAGE_BBOX结构是，counter + bboxes ...
            output_array_device.resize(max_batch_size, 1 + MAX_IMAGE_BBOX * NUM_BOX_ELEMENT).to_gpu();

            vector<Job> fetch_jobs;
            while(get_jobs_and_wait(fetch_jobs, max_batch_size)){

                int infer_batch_size = fetch_jobs.size();
                input->resize_single_dim(0, infer_batch_size);

                for(int ibatch = 0; ibatch < infer_batch_size; ++ibatch){
                    auto& job  = fetch_jobs[ibatch];
                    auto& mono = job.mono_tensor->data();
                    affin_matrix_device.copy_from_gpu(affin_matrix_device.offset(ibatch), mono->get_workspace()->gpu(), 6);
                    input->copy_from_gpu(input->offset(ibatch), mono->gpu(), mono->count());
                    job.mono_tensor->release();
                }

                engine->forward(false);
                output_array_device.to_gpu(false);
                for(int ibatch = 0; ibatch < infer_batch_size; ++ibatch){
                    
                    auto& job                 = fetch_jobs[ibatch];
                    float* image_based_output = output->gpu<float>(ibatch);
                    float* output_array_ptr   = output_array_device.gpu<float>(ibatch);
                    auto affine_matrix        = affin_matrix_device.gpu<float>(ibatch);
                    checkCudaRuntime(hipMemsetAsync(output_array_ptr, 0, sizeof(int), stream_));
                    decode_kernel_invoker(image_based_output, output->size(1), num_classes, confidence_threshold_, nms_threshold_, affine_matrix, output_array_ptr, MAX_IMAGE_BBOX, stream_);
                }

                output_array_device.to_cpu();
                for(int ibatch = 0; ibatch < infer_batch_size; ++ibatch){
                    float* parray = output_array_device.cpu<float>(ibatch);
                    int count     = min(MAX_IMAGE_BBOX, (int)*parray);
                    auto& job     = fetch_jobs[ibatch];
                    auto& image_based_boxes   = job.output;
                    for(int i = 0; i < count; ++i){
                        float* pbox  = parray + 1 + i * NUM_BOX_ELEMENT;
                        int label    = pbox[5];
                        int keepflag = pbox[6];
                        if(keepflag == 1){
                            image_based_boxes.emplace_back(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], label);
                        }
                    }
                    job.pro->set_value(image_based_boxes);
                }
                fetch_jobs.clear();
            }
            stream_ = nullptr;
            tensor_allocator_.reset();
            INFO("Engine destroy.");
        }

        virtual bool preprocess(Job& job, const Mat& image) override{

            if(tensor_allocator_ == nullptr){
                INFOE("tensor_allocator_ is nullptr");
                return false;
            }

            job.mono_tensor = tensor_allocator_->query();
            if(job.mono_tensor == nullptr){
                INFOE("Tensor allocator query failed.");
                return false;
            }

            AutoDevice auto_device(gpu_);
            auto& tensor = job.mono_tensor->data();
            if(tensor == nullptr){
                // not init
                tensor = make_shared<Tensor>();
                tensor->set_workspace(make_shared<MixMemory>());
            }

            Size input_size(input_width_, input_height_);
            job.additional.compute(image.size(), input_size);
            
            tensor->set_stream(stream_);
            tensor->resize(1, 3, input_height_, input_width_);

            size_t size_image      = image.cols * image.rows * 3;
            size_t size_matrix     = upbound(sizeof(job.additional.d2i), 32);
            auto workspace         = tensor->get_workspace();
            uint8_t* gpu_workspace        = (uint8_t*)workspace->gpu(size_matrix + size_image);
            float*   affine_matrix_device = (float*)gpu_workspace;
            uint8_t* image_device         = size_matrix + gpu_workspace;

            uint8_t* cpu_workspace        = (uint8_t*)workspace->cpu(size_matrix + size_image);
            float* affine_matrix_host     = (float*)cpu_workspace;
            uint8_t* image_host           = size_matrix + cpu_workspace;

            //checkCudaRuntime(hipMemcpyAsync(image_host,   image.data, size_image, hipMemcpyHostToHost,   stream_));
            // speed up
            memcpy(image_host, image.data, size_image);
            memcpy(affine_matrix_host, job.additional.d2i, sizeof(job.additional.d2i));
            checkCudaRuntime(hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream_));
            checkCudaRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(job.additional.d2i), hipMemcpyHostToDevice, stream_));

            warp_affine_bilinear_and_normalize_plane(
                image_device,         image.cols * 3,       image.cols,       image.rows, 
                tensor->gpu<float>(), input_width_,         input_height_, 
                affine_matrix_device, 114, 
                normalize_, stream_
            );
            return true;
        }

        virtual vector<shared_future<BoxArray>> commits(const vector<Mat>& images) override{
            return ThreadSafedAsyncInferImpl::commits(images);
        }

        virtual std::shared_future<BoxArray> commit(const Mat& image) override{
            return ThreadSafedAsyncInferImpl::commit(image);
        }

    private:
        int input_width_            = 0;
        int input_height_           = 0;
        int gpu_                    = 0;
        float confidence_threshold_ = 0;
        float nms_threshold_        = 0;
        hipStream_t stream_       = nullptr;
        Norm normalize_;
    };

    void image_to_tensor(const cv::Mat& image, shared_ptr<Tensor>& tensor, Type type, int ibatch){

        Norm normalize;
        if(type == Type::V5){
            normalize = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
        }else if(type == Type::X){
            //float mean[] = {0.485, 0.456, 0.406};
            //float std[]  = {0.229, 0.224, 0.225};
            //normalize_ = CUDAKernel::Norm::mean_std(mean, std, 1/255.0f, CUDAKernel::ChannelType::Invert);
            normalize = Norm::None();
        }else{
            INFOE("Unsupport type %d", type);
        }
        
        Size input_size(tensor->size(3), tensor->size(2));
        AffineMatrix affine;
        affine.compute(image.size(), input_size);

        size_t size_image      = image.cols * image.rows * 3;
        size_t size_matrix     = upbound(sizeof(affine.d2i), 32);
        auto workspace         = tensor->get_workspace();
        uint8_t* gpu_workspace        = (uint8_t*)workspace->gpu(size_matrix + size_image);
        float*   affine_matrix_device = (float*)gpu_workspace;
        uint8_t* image_device         = size_matrix + gpu_workspace;

        uint8_t* cpu_workspace        = (uint8_t*)workspace->cpu(size_matrix + size_image);
        float* affine_matrix_host     = (float*)cpu_workspace;
        uint8_t* image_host           = size_matrix + cpu_workspace;
        auto stream                   = tensor->get_stream();

        memcpy(image_host, image.data, size_image);
        memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
        checkCudaRuntime(hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream));
        checkCudaRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(affine.d2i), hipMemcpyHostToDevice, stream));

        warp_affine_bilinear_and_normalize_plane(
            image_device,               image.cols * 3,       image.cols,       image.rows, 
            tensor->gpu<float>(ibatch), input_size.width,     input_size.height, 
            affine_matrix_device, 114, 
            normalize, stream
        );
    }

    shared_ptr<Infer> create_infer(const string& engine_file, Type type, int gpuid, float confidence_threshold, float nms_threshold){
        shared_ptr<YoloTRTInferImpl> instance(new YoloTRTInferImpl());
        if(!instance->startup(engine_file, type, gpuid, confidence_threshold, nms_threshold)){
            instance.reset();
        }
        return instance;
    }

    //////////////////////////////////////Compile Model/////////////////////////////////////////////////////////////

    const char* mode_string(Mode type) {
        switch (type) {
        case Mode::FP32:
            return "FP32";
        case Mode::FP16:
            return "FP16";
        case Mode::INT8:
            return "INT8";
        default:
            return "UnknowCompileMode";
        }
    }

    typedef std::function<void(int current, int count, const std::vector<std::string>& files, std::shared_ptr<Tensor>& tensor)> Int8Process;

    class Int8EntropyCalibrator : public IInt8EntropyCalibrator2{
    public:
        Int8EntropyCalibrator(const vector<string>& imagefiles, nvinfer1::Dims dims, const Int8Process& preprocess) {

            Assert(preprocess != nullptr);
            this->dims_ = dims;
            this->allimgs_ = imagefiles;
            this->preprocess_ = preprocess;
            this->fromCalibratorData_ = false;
            files_.resize(dims.d[0]);
            checkCudaRuntime(hipStreamCreate(&stream_));
        }

        Int8EntropyCalibrator(const vector<uint8_t>& entropyCalibratorData, nvinfer1::Dims dims, const Int8Process& preprocess) {
            Assert(preprocess != nullptr);

            this->dims_ = dims;
            this->entropyCalibratorData_ = entropyCalibratorData;
            this->preprocess_ = preprocess;
            this->fromCalibratorData_ = true;
            files_.resize(dims.d[0]);
            checkCudaRuntime(hipStreamCreate(&stream_));
        }

        virtual ~Int8EntropyCalibrator(){
            checkCudaRuntime(hipStreamDestroy(stream_));
        }

        int getBatchSize() const noexcept {
            return dims_.d[0];
        }

        bool next() {
            int batch_size = dims_.d[0];
            if (cursor_ + batch_size > allimgs_.size())
                return false;

            int old_cursor = cursor_;
            for(int i = 0; i < batch_size; ++i)
                files_[i] = allimgs_[cursor_++];

            if (!tensor_){
                tensor_.reset(new Tensor(dims_.nbDims, dims_.d));
                tensor_->set_stream(stream_);
                tensor_->set_workspace(make_shared<MixMemory>());
            }

            preprocess_(old_cursor, allimgs_.size(), files_, tensor_);
            return true;
        }

        bool getBatch(void* bindings[], const char* names[], int nbBindings) noexcept {
            if (!next()) return false;
            bindings[0] = tensor_->gpu();
            return true;
        }

        const vector<uint8_t>& getEntropyCalibratorData() {
            return entropyCalibratorData_;
        }

        const void* readCalibrationCache(size_t& length) noexcept {
            if (fromCalibratorData_) {
                length = this->entropyCalibratorData_.size();
                return this->entropyCalibratorData_.data();
            }

            length = 0;
            return nullptr;
        }

        virtual void writeCalibrationCache(const void* cache, size_t length) noexcept {
            entropyCalibratorData_.assign((uint8_t*)cache, (uint8_t*)cache + length);
        }

    private:
        Int8Process preprocess_;
        vector<string> allimgs_;
        size_t batchCudaSize_ = 0;
        int cursor_ = 0;
        nvinfer1::Dims dims_;
        vector<string> files_;
        shared_ptr<Tensor> tensor_;
        vector<uint8_t> entropyCalibratorData_;
        bool fromCalibratorData_ = false;
        hipStream_t stream_ = nullptr;
    };

    bool compile(
        Mode mode, Type type,
        unsigned int max_batch_size,
        const string& source_onnx,
        const string& saveto,
        size_t max_workspace_size,
        const std::string& int8_images_folder,
        const std::string& int8_entropy_calibrator_cache_file) {

        bool hasEntropyCalibrator = false;
        vector<uint8_t> entropyCalibratorData;
        vector<string> entropyCalibratorFiles;

        auto int8process = [=](int current, int count, const vector<string>& files, shared_ptr<Tensor>& tensor){

            for(int i = 0; i < files.size(); ++i){

                auto& file = files[i];
                INFO("Int8 load %d / %d, %s", current + i + 1, count, file.c_str());

                auto image = cv::imread(file);
                if(image.empty()){
                    INFOE("Load image failed, %s", file.c_str());
                    continue;
                }
                image_to_tensor(image, tensor, type, i);
            }
            tensor->synchronize();
        };

        if (mode == Mode::INT8) {
            if (!int8_entropy_calibrator_cache_file.empty()) {
                if (exists(int8_entropy_calibrator_cache_file)) {
                    entropyCalibratorData = load_file(int8_entropy_calibrator_cache_file);
                    if (entropyCalibratorData.empty()) {
                        INFOE("entropyCalibratorFile is set as: %s, but we read is empty.", int8_entropy_calibrator_cache_file.c_str());
                        return false;
                    }
                    hasEntropyCalibrator = true;
                }
            }
            
            if (hasEntropyCalibrator) {
                if (!int8_images_folder.empty()) {
                    INFOW("int8_images_folder is ignore, when int8_entropy_calibrator_cache_file is set");
                }
            }
            else {
                entropyCalibratorFiles = glob_image_files(int8_images_folder);
                if (entropyCalibratorFiles.empty()) {
                    INFOE("Can not find any images(jpg/png/bmp/jpeg/tiff) from directory: %s", int8_images_folder.c_str());
                    return false;
                }

                if(entropyCalibratorFiles.size() < max_batch_size){
                    INFOW("Too few images provided, %d[provided] < %d[max batch size], image copy will be performed", entropyCalibratorFiles.size(), max_batch_size);

                    int old_size = entropyCalibratorFiles.size();
                    for(int i = old_size; i < max_batch_size; ++i)
                        entropyCalibratorFiles.push_back(entropyCalibratorFiles[i % old_size]);
                }
            }
        }
        else {
            if (hasEntropyCalibrator) {
                INFOW("int8_entropy_calibrator_cache_file is ignore, when Mode is '%s'", mode_string(mode));
            }
        }

        INFO("Compile %s %s.", mode_string(mode), source_onnx.c_str());
        shared_ptr<IBuilder> builder(createInferBuilder(gLogger), destroy_nvidia_pointer<IBuilder>);
        if (builder == nullptr) {
            INFOE("Can not create builder.");
            return false;
        }

        shared_ptr<IBuilderConfig> config(builder->createBuilderConfig(), destroy_nvidia_pointer<IBuilderConfig>);
        if (mode == Mode::FP16) {
            if (!builder->platformHasFastFp16()) {
                INFOW("Platform not have fast fp16 support");
            }
            config->setFlag(BuilderFlag::kFP16);
        }
        else if (mode == Mode::INT8) {
            if (!builder->platformHasFastInt8()) {
                INFOW("Platform not have fast int8 support");
            }
            config->setFlag(BuilderFlag::kINT8);
        }

        shared_ptr<INetworkDefinition> network;
        shared_ptr<nvonnxparser::IParser> onnxParser;
        const auto explicitBatch = 1U << static_cast<uint32_t>(nvinfer1::NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
        network = shared_ptr<INetworkDefinition>(builder->createNetworkV2(explicitBatch), destroy_nvidia_pointer<INetworkDefinition>);

        //from onnx is not markOutput
        onnxParser.reset(nvonnxparser::createParser(*network, gLogger), destroy_nvidia_pointer<nvonnxparser::IParser>);
        if (onnxParser == nullptr) {
            INFOE("Can not create parser.");
            return false;
        }

        if (!onnxParser->parseFromFile(source_onnx.c_str(), 1)) {
            INFOE("Can not parse OnnX file: %s", source_onnx.c_str());
            return false;
        }
        
        auto inputTensor = network->getInput(0);
        auto inputDims = inputTensor->getDimensions();

        shared_ptr<Int8EntropyCalibrator> int8Calibrator;
        if (mode == Mode::INT8) {
            auto calibratorDims = inputDims;
            calibratorDims.d[0] = max_batch_size;

            if (hasEntropyCalibrator) {
                INFO("Using exist entropy calibrator data[%d bytes]: %s", entropyCalibratorData.size(), int8_entropy_calibrator_cache_file.c_str());
                int8Calibrator.reset(new Int8EntropyCalibrator(
                    entropyCalibratorData, calibratorDims, int8process
                ));
            }
            else {
                INFO("Using image list[%d files]: %s", entropyCalibratorFiles.size(), int8_images_folder.c_str());
                int8Calibrator.reset(new Int8EntropyCalibrator(
                    entropyCalibratorFiles, calibratorDims, int8process
                ));
            }
            config->setInt8Calibrator(int8Calibrator.get());
        }

        INFO("Input shape is %s", join_dims(vector<int>(inputDims.d, inputDims.d + inputDims.nbDims)).c_str());
        INFO("Set max batch size = %d", max_batch_size);
        INFO("Set max workspace size = %.2f MB", max_workspace_size / 1024.0f / 1024.0f);

        int net_num_input = network->getNbInputs();
        INFO("Network has %d inputs:", net_num_input);
        vector<string> input_names(net_num_input);
        for(int i = 0; i < net_num_input; ++i){
            auto tensor = network->getInput(i);
            auto dims = tensor->getDimensions();
            auto dims_str = join_dims(vector<int>(dims.d, dims.d+dims.nbDims));
            INFO("      %d.[%s] shape is %s", i, tensor->getName(), dims_str.c_str());

            input_names[i] = tensor->getName();
        }

        int net_num_output = network->getNbOutputs();
        INFO("Network has %d outputs:", net_num_output);
        for(int i = 0; i < net_num_output; ++i){
            auto tensor = network->getOutput(i);
            auto dims = tensor->getDimensions();
            auto dims_str = join_dims(vector<int>(dims.d, dims.d+dims.nbDims));
            INFO("      %d.[%s] shape is %s", i, tensor->getName(), dims_str.c_str());
        }

        int net_num_layers = network->getNbLayers();
        INFO("Network has %d layers", net_num_layers);		
        builder->setMaxBatchSize(max_batch_size);
        config->setMaxWorkspaceSize(max_workspace_size);

        auto profile = builder->createOptimizationProfile();
        for(int i = 0; i < net_num_input; ++i){
            auto input = network->getInput(i);
            auto input_dims = input->getDimensions();
            input_dims.d[0] = 1;
            profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMIN, input_dims);
            profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kOPT, input_dims);
            input_dims.d[0] = max_batch_size;
            profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMAX, input_dims);
        }
        config->addOptimizationProfile(profile);

        INFO("Building engine...");
        auto time_start = chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count();
        shared_ptr<ICudaEngine> engine(builder->buildEngineWithConfig(*network, *config), destroy_nvidia_pointer<ICudaEngine>);
        if (engine == nullptr) {
            INFOE("engine is nullptr");
            return false;
        }

        if (mode == Mode::INT8) {
            if (!hasEntropyCalibrator) {
                if (!int8_entropy_calibrator_cache_file.empty()) {
                    INFO("Save calibrator to: %s", int8_entropy_calibrator_cache_file.c_str());
                    save_file(int8_entropy_calibrator_cache_file, int8Calibrator->getEntropyCalibratorData());
                }
                else {
                    INFO("No set entropyCalibratorFile, and entropyCalibrator will not save.");
                }
            }
        }

        auto time_end = chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count();
        INFO("Build done %lld ms !", time_end - time_start);
        
        // serialize the engine, then close everything down
        shared_ptr<IHostMemory> seridata(engine->serialize(), destroy_nvidia_pointer<IHostMemory>);
        return save_file(saveto, seridata->data(), seridata->size());
    }
};