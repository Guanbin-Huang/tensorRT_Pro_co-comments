#include "hip/hip_runtime.h"

#include "simple_yolo.hpp"
#include <NvInfer.h>
#include <NvOnnxParser.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <fstream>
#include <memory>
#include <string>
#include <future>
#include <condition_variable>
#include <mutex>
#include <thread>
#include <queue>

#if defined(_WIN32)
#	include <Windows.h>
#   include <wingdi.h>
#	include <Shlwapi.h>
#	pragma comment(lib, "shlwapi.lib")
#	undef min
#	undef max
#else
#	include <dirent.h>
#	include <sys/types.h>
#	include <sys/stat.h>
#	include <unistd.h>
#   include <stdarg.h>
#endif

namespace SimpleYolo{

    using namespace nvinfer1;
    using namespace std;
    using namespace cv;

    #define CURRENT_DEVICE_ID   -1
    #define GPU_BLOCK_THREADS  512
    #define KernelPositionBlock											\
        int position = (blockDim.x * blockIdx.x + threadIdx.x);		    \
        if (position >= (edge)) return;

    #define checkCudaRuntime(call) check_runtime(call, #call, __LINE__, __FILE__)
    static bool check_runtime(hipError_t e, const char* call, int line, const char *file);

    #define checkCudaKernel(...)                                                                         \
        __VA_ARGS__;                                                                                     \
        do{hipError_t cudaStatus = hipPeekAtLastError();                                               \
        if (cudaStatus != hipSuccess){                                                                  \
            INFOE("launch failed: %s", hipGetErrorString(cudaStatus));                                  \
        }} while(0);

    #define Assert(op)					 \
        do{                              \
            bool cond = !(!(op));        \
            if(!cond){                   \
                INFOF("Assert failed, " #op);  \
            }                                  \
        }while(false)

    /* 修改这个level来实现修改日志输出级别 */
    #define CURRENT_LOG_LEVEL       LogLevel::Info
    #define INFOD(...)			__log_func(__FILE__, __LINE__, LogLevel::Debug, __VA_ARGS__)
    #define INFOV(...)			__log_func(__FILE__, __LINE__, LogLevel::Verbose, __VA_ARGS__)
    #define INFO(...)			__log_func(__FILE__, __LINE__, LogLevel::Info, __VA_ARGS__)
    #define INFOW(...)			__log_func(__FILE__, __LINE__, LogLevel::Warning, __VA_ARGS__)
    #define INFOE(...)			__log_func(__FILE__, __LINE__, LogLevel::Error, __VA_ARGS__)
    #define INFOF(...)			__log_func(__FILE__, __LINE__, LogLevel::Fatal, __VA_ARGS__)

    enum class NormType : int{
        None      = 0,
        MeanStd   = 1,
        AlphaBeta = 2
    };

    enum class ChannelType : int{
        None          = 0,
        SwapRB        = 1
    };

    /* 归一化操作，可以支持均值标准差，alpha beta，和swap RB */
    struct Norm{
        float mean[3];
        float std[3];
        float alpha, beta;
        NormType type = NormType::None;
        ChannelType channel_type = ChannelType::None;

        // out = (x * alpha - mean) / std
        static Norm mean_std(const float mean[3], const float std[3], float alpha = 1/255.0f, ChannelType channel_type=ChannelType::None);

        // out = x * alpha + beta
        static Norm alpha_beta(float alpha, float beta = 0, ChannelType channel_type=ChannelType::None);

        // None
        static Norm None();
    };

    Norm Norm::mean_std(const float mean[3], const float std[3], float alpha, ChannelType channel_type){

        Norm out;
        out.type  = NormType::MeanStd;
        out.alpha = alpha;
        out.channel_type = channel_type;
        memcpy(out.mean, mean, sizeof(out.mean));
        memcpy(out.std,  std,  sizeof(out.std));
        return out;
    }

    Norm Norm::alpha_beta(float alpha, float beta, ChannelType channel_type){

        Norm out;
        out.type = NormType::AlphaBeta;
        out.alpha = alpha;
        out.beta = beta;
        out.channel_type = channel_type;
        return out;
    }

    Norm Norm::None(){
        return Norm();
    }

    /* 构造时设置当前gpuid，析构时修改为原来的gpuid */
    class AutoDevice{
    public:
        AutoDevice(int device_id = 0){
            hipGetDevice(&old_);
                checkCudaRuntime(hipSetDevice(device_id));
        }

        virtual ~AutoDevice(){
                checkCudaRuntime(hipSetDevice(old_));
        }
    
    private:
        int old_ = -1;
    };
    
    enum class LogLevel : int{
        Debug   = 5,
        Verbose = 4,
        Info    = 3,
        Warning = 2,
        Error   = 1,
        Fatal   = 0
    };

    static void __log_func(const char* file, int line, LogLevel level, const char* fmt, ...);
    inline int upbound(int n, int align = 32){return (n + align - 1) / align * align;}

    static bool check_runtime(hipError_t e, const char* call, int line, const char *file){
        if (e != hipSuccess) {
            INFOE("CUDA Runtime error %s # %s, code = %s [ %d ] in file %s:%d", call, hipGetErrorString(e), hipGetErrorName(e), e, file, line);
            return false;
        }
        return true;
    }

    #define TRT_STR(v)  #v
    #define TRT_VERSION_STRING(major, minor, patch, build)   TRT_STR(major) "." TRT_STR(minor) "." TRT_STR(patch) "." TRT_STR(build)
    const char* trt_version(){
        return TRT_VERSION_STRING(NV_TENSORRT_MAJOR, NV_TENSORRT_MINOR, NV_TENSORRT_PATCH, NV_TENSORRT_BUILD);
    }

    static bool check_device_id(int device_id){
        int device_count = -1;
        checkCudaRuntime(hipGetDeviceCount(&device_count));
        if(device_id < 0 || device_id >= device_count){
            INFOE("Invalid device id: %d, count = %d", device_id, device_count);
            return false;
        }
        return true;
    }

    static bool exists(const string& path){

    #ifdef _WIN32
        return ::PathFileExistsA(path.c_str());
    #else
        return access(path.c_str(), R_OK) == 0;
    #endif
    }

    static const char* level_string(LogLevel level){
        switch (level){
            case LogLevel::Debug: return "debug";
            case LogLevel::Verbose: return "verbo";
            case LogLevel::Info: return "info";
            case LogLevel::Warning: return "warn";
            case LogLevel::Error: return "error";
            case LogLevel::Fatal: return "fatal";
            default: return "unknow";
        }
    }

    template<typename _T>
    static string join_dims(const vector<_T>& dims){
        stringstream output;
        char buf[64];
        const char* fmts[] = {"%d", " x %d"};
        for(int i = 0; i < dims.size(); ++i){
            snprintf(buf, sizeof(buf), fmts[i != 0], dims[i]);
            output << buf;
        }
        return output.str();
    }

    static bool save_file(const string& file, const void* data, size_t length){

        FILE* f = fopen(file.c_str(), "wb");
        if (!f) return false;

        if (data && length > 0){
            if (fwrite(data, 1, length, f) != length){
                fclose(f);
                return false;
            }
        }
        fclose(f);
        return true;
    }

    static bool save_file(const string& file, const vector<uint8_t>& data){
        return save_file(file, data.data(), data.size());
    }

    static string file_name(const string& path, bool include_suffix){

        if (path.empty()) return "";

        int p = path.rfind('/');

#ifdef U_OS_WINDOWS
        int e = path.rfind('\\');
        p = std::max(p, e);
#endif
        p += 1;

        //include suffix
        if (include_suffix)
            return path.substr(p);

        int u = path.rfind('.');
        if (u == -1)
            return path.substr(p);

        if (u <= p) u = path.size();
        return path.substr(p, u - p);
    }

    /*  遍历文件夹图片  */
    vector<string> glob_image_files(const string& directory){

        /* 检索目录下的所有图像："*.jpg;*.png;*.bmp;*.jpeg;*.tiff" */
        vector<string> files, output;
        set<string> pattern_set{"jpg", "png", "bmp", "jpeg", "tiff"};

        if(directory.empty()){
            INFOE("Glob images from folder failed, folder is empty");
            return output;
        }

        try{
			vector<cv::String> files_;
			files_.reserve(10000);
            cv::glob(directory + "/*", files_, true);
			files.insert(files.end(), files_.begin(), files_.end());
        }catch(...){
            INFOE("Glob %s failed", directory.c_str());
            return output;
        }

        for(int i = 0; i < files.size(); ++i){
            auto& file = files[i];
            int p = file.rfind(".");
            if(p == -1) continue;

            auto suffix = file.substr(p+1);
            std::transform(suffix.begin(), suffix.end(), suffix.begin(), [](char c){
                if(c >= 'A' && c <= 'Z')
                    c -= 'A' + 'a';
                return c;
            });
            if(pattern_set.find(suffix) != pattern_set.end())
                output.push_back(file);
        }
        return output;
    }

    static void __log_func(const char* file, int line, LogLevel level, const char* fmt, ...){

        if(level > CURRENT_LOG_LEVEL)
            return;

        va_list vl;
        va_start(vl, fmt);
        
        char buffer[2048];
        string filename = file_name(file, true);
        int n = snprintf(buffer, sizeof(buffer), "[%s][%s:%d]:", level_string(level), filename.c_str(), line);
        vsnprintf(buffer + n, sizeof(buffer) - n, fmt, vl);

        fprintf(stdout, "%s\n", buffer);
        if (level == LogLevel::Fatal) {
            fflush(stdout);
            abort();
        }
    }

    static dim3 grid_dims(int numJobs) {
        int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
        return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
    }

    static dim3 block_dims(int numJobs) {
        return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
    }

    static int get_device(int device_id){
        if(device_id != CURRENT_DEVICE_ID){
            check_device_id(device_id);
            return device_id;
        }

        checkCudaRuntime(hipGetDevice(&device_id));
        return device_id;
    }

    void set_device(int device_id) {
        if (device_id == -1)
            return;

        checkCudaRuntime(hipSetDevice(device_id));
    }

    /////////////////////////////CUDA kernels////////////////////////////////////////////////

    const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
    static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
        *ox = matrix[0] * x + matrix[1] * y + matrix[2];
        *oy = matrix[3] * x + matrix[4] * y + matrix[5];
    }
    
    /* 解码核函数 */
    static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects){  
   
        /* 这里需要主要的是传入的参数num_bboxes就是25200, 这是3个head的输出concat的，如下：
        *   B × 3 × 85 × 80 × 80  --> B × 3 × 80 × 80 × 85   --> B × 19200 × 85
            B × 3 × 85 × 40 × 40  --> B × 3 × 40 × 40 × 85   --> B × 4800 × 85   ----> B × 25200 × 85
            B × 3 × 85 × 20 × 20  --> B × 3 × 20 × 20 × 85   --> B × 1200 × 85
            由此可以看出就是我们onnx导出的输出，25200分别是3个head的concat，每一个就是特征图的点，这个一定要理解，
            对应的是特征图二维的每个位置，，存储的方式是一维的，因此取数据就需要通过计算获取数据,这里B此时为1
        */
  
        /* 开启25600个线程进行加速，但是实际只需要25200个线程进行加速处理 */
        int position = blockDim.x * blockIdx.x + threadIdx.x;
        
        if (position >= num_bboxes) return;
        /* 
            这里应该很容易理解了，因为数据是 1 × 25200 × 85，在数据存储时是顺序存储的， 其中前25200个数据是开启的并行线程，也就是此时的25200是同时开始处理，
            后面跟的就是对应的85个数据，但是这85个数据是进行一维数组存储的， 因此想要分别查找到对应的85就需要每个线程乘上85就可以找到
            对应的起点了，好好思考
        */
        float* pitem     = predict + (5 + num_classes) * position;
        /*
            获取到每个线程对应点的85(5+80)数据起始位置后,分别进行提取对应的数据，objectness为对象obj置信度
        */
        float objectness = pitem[4];
        /* 如果小于设置的obj 置信度阈值，该线程返回 */
        if(objectness < confidence_threshold)
            return;
        /* 在后面class_confidence就是类别的置信度，因为是80类，因此循环80次 */
        float* class_confidence = pitem + 5;
        float confidence        = *class_confidence++;
        int label               = 0;
        /* for循环的目的是获取80类中概率最大的那个类别 */
        for(int i = 1; i < num_classes; ++i, ++class_confidence){
            if(*class_confidence > confidence){
                confidence = *class_confidence;
                label      = i;
            }
        }
        /* 这个就是训练时损失有两个置信度相乘，这里也体现了一个是obj置信度另一个是类别置信度 */
        confidence *= objectness;
        /* 如果总的置信度还是小于阈值，直接返回 */
        if(confidence < confidence_threshold)
            return;
        /* 反之说明该预测有效，需要保留相关数据 */
        int index = atomicAdd(parray, 1);
        if(index >= max_objects)
            return;
        /* 提取当前的85的前4个数据， 其实就是cx，cy，width，height */
        float cx         = *pitem++;
        float cy         = *pitem++;
        float width      = *pitem++;
        float height     = *pitem++;
        /* 同时转换为左上角坐标点和右下角坐标点 */
        float left   = cx - width * 0.5f;
        float top    = cy - height * 0.5f;
        float right  = cx + width * 0.5f;
        float bottom = cy + height * 0.5f;
        /* 下面进行仿射反变换为原始图片下的坐标 */
        affine_project(invert_affine_matrix, left,  top,    &left,  &top);
        affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

        /* 
        *  NUM_BOX_ELEMENT是限制最多的bbox的大小
        */
        float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
        *pout_item++ = left;
        *pout_item++ = top;
        *pout_item++ = right;
        *pout_item++ = bottom;
        *pout_item++ = confidence;
        *pout_item++ = label;
        *pout_item++ = 1; // 1 = keep, 0 = ignore
    }

    static __device__ float box_iou(
        float aleft, float atop, float aright, float abottom, 
        float bleft, float btop, float bright, float bbottom
    ){

        float cleft 	= max(aleft, bleft);
        float ctop 		= max(atop, btop);
        float cright 	= min(aright, bright);
        float cbottom 	= min(abottom, bbottom);
        
        float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
        if(c_area == 0.0f)
            return 0.0f;
        
        float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
        float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
        return c_area / (a_area + b_area - c_area);
    }
    
    static __global__ void fast_nms_kernel(float* bboxes, int max_objects, float threshold){

        /* 开启的线程数最大为1024， 但是实际存在小于1024的情况，因此如下处理 */
        int position = (blockDim.x * blockIdx.x + threadIdx.x);
        /* 去线程数和实际bbox的最小值 */
        int count = min((int)*bboxes, max_objects);
        if (position >= count) 
            return;
        
        /* 正常情况下，数组应该从0开始索引，但是因为存储时是float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
        *  因此取数据时也要这样取，先取出一组数据为pcurrent，拿这个和其他的bbox比较，
           如果置信度大于当前值，就需要进行通过iou进行判定
        */
        // left, top, right, bottom, confidence, class, keepflag
        float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
        for(int i = 0; i < count; ++i){
            float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
            /* 如果对比的是同一组数据或者不同类数据，跳过当前的bbox */
            if(i == position || pcurrent[5] != pitem[5]) continue;
            /* 反之处理的不是同一个bbox， 继续向下处理，如果pitem的置信度大于当前的置信度，则继续处理，反之跳过 */
            if(pitem[4] >= pcurrent[4]){
                /* 如果置信度相同，直接跳过 */
                if(pitem[4] == pcurrent[4] && i < position)
                    continue;
                /* 如果置信度大于当前的置信度，则进一步通过iou进行处理  */
                float iou = box_iou(
                    pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                    pitem[0],    pitem[1],    pitem[2],    pitem[3]
                );
                /* 如果计算出来的iou大于阈值，则当前的bbox失效，反之保持 */
                if(iou > threshold){
                    pcurrent[6] = 0;  // 1=keep, 0=ignore
                    return;
                }
            }
            /* 最终通过bboxes[6]的状态进行确定即可 */
        }
    } 

    static void decode_kernel_invoker(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float nms_threshold, float* invert_affine_matrix, float* parray, int max_objects, hipStream_t stream){
        
        /* 这里需要主要的是传入的参数num_bboxes就是25200, 这是3个head的输出concat的，如下：
        *   B × 3 × 85 × 80 × 80  --> B × 3 × 80 × 80 × 85   --> B × 19200 × 85
            B × 3 × 85 × 40 × 40  --> B × 3 × 40 × 40 × 85   --> B × 4800 × 85   ----> B × 25200 × 85
            B × 3 × 85 × 20 × 20  --> B × 3 × 20 × 20 × 85   --> B × 1200 × 85
            由此可以看出就是我们onnx导出的输出，25200分别是3个head的concat，每一个就是特征图的点，这里需要，这个一定要理解，对应的是特征图二维的每个位置，，存储的方式是一维的，因此取数据就需要
            通过计算获取数据
        */

        auto grid = grid_dims(num_bboxes);
        auto block = block_dims(num_bboxes);
        /* 通过上面的分析可以发现，其每个位置都需要计算，因此需要开辟25200个线程 */
        /* 如果核函数有波浪线，没关系，他是正常的，你只是看不顺眼罢了，下面进入解码核函数  */
        checkCudaKernel(decode_kernel<<<grid, block, 0, stream>>>(predict, num_bboxes, num_classes, confidence_threshold, invert_affine_matrix, parray, max_objects));
        /* 进行非极大值抑制，因为解码中最多输出1024个bbox，因此只需要开启最大的线程数为1024即可 */
        grid = grid_dims(max_objects);
        block = block_dims(max_objects);
        checkCudaKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold));
    }

    /*  数据预处理  */
    static __global__ void warp_affine_bilinear_and_normalize_plane_kernel(uint8_t* src, int src_line_size, int src_width, int src_height, float* dst, int dst_width, int dst_height, 
        uint8_t const_value_st, float* warp_affine_matrix_2_3, Norm norm, int edge){

        /*    这里的warpaffine和python实现原理相同，不同的是这里的实现是通过cuda多线程实现，具体实现原理如下
        *     这里需要确定的是这里为了尽量降低计算量，采用遍历目标图片的像素，显然目标图片的像素大小是确定的，无论输入的图片大小是多大
              最后我都会变换到目标图片大小，如输入到深度学习模型的图片应该是640x640，原始图片的大小为1080x1920，显然遍历原始图片的计算很大
              遍历目标的图片是固定的且不大，那么这个仿射变换如何做呢？
              1. 首先输入的仿射变换矩阵是从原始图片的点--->目标图片的点，因此需要取逆变换获取到从目标图像的点--->原始图片的点
              2. 当变换到原始图片的点位置时，将采用双线性变换的方法计算该点在原始位置的像素值
              3. 如何计算呢？这里需要想明白，双线的本质是通过四个点的值计算一个点的值，那么变换到原始图片的点就是我们需要求的点值，
                 这个计算出来的值将直接赋值到目标图片对应位置，但是这四个点如何选取？其实很简单，就取相邻的四个点即可如：
                  (0,0) (1,0)      (250,250)  (251,250)
                  (0,1) (1,1)      (250,251)  (251,251)
                 这个四个点的选取就是变换过来的点的相邻四个点即可，如何做呢？上下取整即可如上面我举例两个点，
                 假如从目标的点变换到原始图片的点为(250.35，250.65)，那么这个点正好在上面的四个点的范围内，计算相对位置就是(0.35,0.65)
                 然后通过双线性计算该点的值，把该点的值直接赋值目标待求点位置即可，理解到这一步基本就完全理解了
            
        */

        /* 这里的理解和python版本理解类似，主要需要考虑的是CUDA的编程，集cuda的多线程代码
           传入的edge就是线程的边界，即是所有任务的所需的线程
        */
        int position = blockDim.x * blockIdx.x + threadIdx.x;
        if (position >= edge) return;
        /* 获取矩阵的相关参数 */
        float m_x1 = warp_affine_matrix_2_3[0];
        float m_y1 = warp_affine_matrix_2_3[1];
        float m_z1 = warp_affine_matrix_2_3[2];
        float m_x2 = warp_affine_matrix_2_3[3];
        float m_y2 = warp_affine_matrix_2_3[4];
        float m_z2 = warp_affine_matrix_2_3[5];
        /* 因为数据的存储是一维的线性存储，因此需要通过计算获取目的图片的宽高界限 */
        int dx      = position % dst_width;
        int dy      = position / dst_width;
        /* 通过目标的点计算得到在原始图片点的位置，需要对其进行源图像和目标图像几何中心的对齐。 
        float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
        float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
        */
        float src_x = m_x1 * dx + m_y1 * dy + m_z1;
        float src_y = m_x2 * dx + m_y2 * dy + m_z2;
        float c0, c1, c2;

        /* 检查边缘情况，如果是边缘，直接赋常数值 */
        if(src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height){
            // out of range
            c0 = const_value_st;
            c1 = const_value_st;
            c2 = const_value_st;
        }else{
            /*  floorf(x)  获取不大于x的最大整数。其实这两就是取原始坐标的相邻的四个点 */
            int y_low = floorf(src_y);
            int x_low = floorf(src_x);
            int y_high = y_low + 1;
            int x_high = x_low + 1;
            /* 下面就是计算插值的代码 */
            uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
            float ly    = src_y - y_low;
            float lx    = src_x - x_low;
            float hy    = 1 - ly;
            float hx    = 1 - lx;
            float w1    = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
            uint8_t* v1 = const_value;
            uint8_t* v2 = const_value;
            uint8_t* v3 = const_value;
            uint8_t* v4 = const_value;
            if(y_low >= 0){
                if (x_low >= 0)
                    v1 = src + y_low * src_line_size + x_low * 3;

                if (x_high < src_width)
                    v2 = src + y_low * src_line_size + x_high * 3;
            }
            
            if(y_high < src_height){
                if (x_low >= 0)
                    v3 = src + y_high * src_line_size + x_low * 3;

                if (x_high < src_width)
                    v4 = src + y_high * src_line_size + x_high * 3;
            }
            /*
            c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
            c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
            c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
            */
            c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
            c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
            c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
        }

        if(norm.channel_type == ChannelType::SwapRB){
            float t = c2;
            c2 = c0;  c0 = t;
        }

        if(norm.type == NormType::MeanStd){
            c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
            c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
            c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
        }else if(norm.type == NormType::AlphaBeta){
            c0 = c0 * norm.alpha + norm.beta;
            c1 = c1 * norm.alpha + norm.beta;
            c2 = c2 * norm.alpha + norm.beta;
        }
        /* 
            这里需要解释的是，因为传入的是float型的指针，同时因为数据的存储是一维的，这里作者把三通道进行分开存储，因此每个通道
           占用的区域大小为area = dst_width * dst_height，后面就是分别把值填进去即可
        */
        int area = dst_width * dst_height;
        float* pdst_c0 = dst + dy * dst_width + dx;
        float* pdst_c1 = pdst_c0 + area;
        float* pdst_c2 = pdst_c1 + area;
        *pdst_c0 = c0;
        *pdst_c1 = c1;
        *pdst_c2 = c2;
    }

    static void warp_affine_bilinear_and_normalize_plane(
        uint8_t* src, int src_line_size, int src_width, int src_height, float* dst, int dst_width, int dst_height,
        float* matrix_2_3, uint8_t const_value, const Norm& norm,
        hipStream_t stream) {
        
        /* 这里传入的jobs其实就是目标图片的宽高的乘积，目的是因为后面需要开启gpu加速，需要开启多线程，多线程的开启个数就是目的图片的宽高乘积 */
        int jobs   = dst_width * dst_height;
        auto grid  = grid_dims(jobs);
        auto block = block_dims(jobs);
        
        checkCudaKernel(warp_affine_bilinear_and_normalize_plane_kernel << <grid, block, 0, stream >> > (
            src, src_line_size,
            src_width, src_height, dst,
            dst_width, dst_height, const_value, matrix_2_3, norm, jobs
        ));
    }


    //////////////////////////////class MixMemory/////////////////////////////////////////////////
    /* gpu/cpu内存管理
        自动对gpu和cpu内存进行分配和释放
        这里的cpu使用的是pinned memory，当对gpu做内存复制时，性能比较好
        因为是hipHostMalloc分配的，因此他与cuda context有关联

        内存分配的重点在于，CPU和GPU可以互相copy和创建，通常情况下，创建一块内存，首先应该具备以下要求：
        1. 知道指向内存的指针
        2. 开辟内存块的大小
        3. GPU内存的id号
        4. 可以直接引用外部内存块
        通过上面我们可以知道，设计类的出发点应该是需要定义几个变量，然后写方法分别实现我们想要的功能如cpu->gpu, gpu->cpu等等操作
        中间要考虑内存的复用，内存copy的性能等细节，这里大神基本都注意到了值得学习
        因此下面的MixMemory类，就需要着重观察私用成员变量：
                        void* cpu_ = nullptr;                            
                        size_t cpu_size_ = 0;
                        bool owner_cpu_ = true;                   
                        int device_id_ = 0;                       
                        void* gpu_ = nullptr;                   
                        size_t gpu_size_ = 0;
                        bool owner_gpu_ = true;

       通过观察私用成员变量的和成员方法可以很快理解MixMemory         

    */
    class MixMemory {
    public:
        /* 构造和析构函数 */
        MixMemory(int device_id = CURRENT_DEVICE_ID);
        MixMemory(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size);
        virtual ~MixMemory();

        /* 申请gpu内存和cpu内存 */
        void* gpu(size_t size);
        void* cpu(size_t size);
        /* 释放内存 */
        void release_gpu();
        void release_cpu();
        void release_all();
        /* 获取所用权 */
        inline bool owner_gpu() const{return owner_gpu_;}
        inline bool owner_cpu() const{return owner_cpu_;}
        /* 获取申请内存的大小 */
        inline size_t cpu_size() const{return cpu_size_;}
        inline size_t gpu_size() const{return gpu_size_;}
        /* 获取设备id */
        inline int device_id() const{return device_id_;}
        /* 获取GPU内存地址 */
        inline void* gpu() const { return gpu_; }

        // Pinned Memory
        inline void* cpu() const { return cpu_; }

        void reference_data(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size);

    private:
        
        /* cpu指针 */
        void* cpu_ = nullptr;
        /* cpu申请空间大小 大小 */
        size_t cpu_size_ = 0;
        bool owner_cpu_ = true;
        /* GPU 0 */
        int device_id_ = 0;
        /* GPU指针 */
        void* gpu_ = nullptr;
        /* GPU申请空间大小 */
        size_t gpu_size_ = 0;
        bool owner_gpu_ = true;
    };

    MixMemory::MixMemory(int device_id){
        device_id_ = get_device(device_id);
    }
    /* 传入CPU地址和GPU地址以及对应的大小对其进行初始化 */
    MixMemory::MixMemory(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size){
        reference_data(cpu, cpu_size, gpu, gpu_size);		
    }
    /* 引用数据 */
    void MixMemory::reference_data(void* cpu, size_t cpu_size, void* gpu, size_t gpu_size){
        release_all();
        
        if(cpu == nullptr || cpu_size == 0){
            cpu = nullptr;
            cpu_size = 0;
        }

        if(gpu == nullptr || gpu_size == 0){
            gpu = nullptr;
            gpu_size = 0;
        }

        /* 把传入进来的参数进行赋值 */
        this->cpu_ = cpu;
        this->cpu_size_ = cpu_size;
        this->gpu_ = gpu;
        this->gpu_size_ = gpu_size;
        
        /* 下面两行代码有什么作用呢？ */
        /* 大神解释： 可以允许MixMemory引用一块内存，不属于自己管理，但是可以引用 */
        this->owner_cpu_ = !(cpu && cpu_size > 0);
        this->owner_gpu_ = !(gpu && gpu_size > 0);
        checkCudaRuntime(hipGetDevice(&device_id_));
    }

    MixMemory::~MixMemory() {
        release_all();
    }

    void* MixMemory::gpu(size_t size) {
        /* 这里判断需要开辟的空间size，和之前的开辟空间的size大小比较，如果小，则直接返回即可
           如果大则需要重新开辟空间，先释放已分配的空间，然后开辟新空间，同时把新空间设置为0
        */
        if (gpu_size_ < size) {
            release_gpu();

            gpu_size_ = size;
            AutoDevice auto_device_exchange(device_id_);
            checkCudaRuntime(hipMalloc(&gpu_, size));
            checkCudaRuntime(hipMemset(gpu_, 0, size));
        }
        return gpu_;
    }

    void* MixMemory::cpu(size_t size) {

        if (cpu_size_ < size) {
            release_cpu();

            cpu_size_ = size;
            AutoDevice auto_device_exchange(device_id_);
            checkCudaRuntime(hipHostMalloc(&cpu_, size));
            Assert(cpu_ != nullptr);
            memset(cpu_, 0, size);
        }
        return cpu_;
    }
    /* 释放CPU资源 */
    void MixMemory::release_cpu() {
        if (cpu_) {
            if(owner_cpu_){
                AutoDevice auto_device_exchange(device_id_);
                checkCudaRuntime(hipHostFree(cpu_));
            }
            cpu_ = nullptr;
        }
        cpu_size_ = 0;
    }

    /* 释放GPU资源 */
    void MixMemory::release_gpu() {
        if (gpu_) {
            if(owner_gpu_){
                AutoDevice auto_device_exchange(device_id_);
                checkCudaRuntime(hipFree(gpu_));
            }
            gpu_ = nullptr;
        }
        gpu_size_ = 0;
    }

    /* 释放所以资源 */
    void MixMemory::release_all() {
        release_cpu();
        release_gpu();
    }

    /////////////////////////////////class Tensor////////////////////////////////////////////////
    /* Tensor类，实现张量的管理
        由于NN多用张量，必须有个类进行管理才方便，实现内存自动分配，计算索引等等
        如果要调试，可以执行save_to_file，储存为文件后，在python中加载并查看
    */
    enum class DataHead : int{
        Init   = 0,
        Device = 1,
        Host   = 2
    };

    class Tensor {
    public:
        Tensor(const Tensor& other) = delete;
        Tensor& operator = (const Tensor& other) = delete;
        /* 构造和析构函数 */
        explicit Tensor(std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        explicit Tensor(int n, int c, int h, int w, std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        explicit Tensor(int ndims, const int* dims, std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        explicit Tensor(const std::vector<int>& dims, std::shared_ptr<MixMemory> data = nullptr, int device_id = CURRENT_DEVICE_ID);
        virtual ~Tensor();

        int numel() const;
        inline int ndims() const{return shape_.size();} /* 获取维度 */
        inline int size(int index)  const{return shape_[index];} /* 获取某一维维度的大小 */
        inline int shape(int index) const{return shape_[index];} /* 获取某一维维度的大小 */
        /*  获取维度的相关信息 */
        inline int batch()   const{return shape_[0];}
        inline int channel() const{return shape_[1];}
        inline int height()  const{return shape_[2];}
        inline int width()   const{return shape_[3];}

        inline const std::vector<int>& dims() const { return shape_; }
        inline int bytes()                    const { return bytes_; }
        inline int bytes(int start_axis)      const { return count(start_axis) * element_size(); }/* 获取数据所占字节数 */
        inline int element_size()             const { return sizeof(float); }
        inline DataHead head()                const { return head_; } /* 判断是GPU数据还是cpu数据还是初始化 */

        std::shared_ptr<Tensor> clone() const;
        Tensor& release(); /* 释放资源 */
        Tensor& set_to(float value);
        bool empty() const; /* 判断数据是否为空 */

        /* tensor的数据偏置索引 */
        template<typename ... _Args>
        int offset(int index, _Args ... index_args) const{
            const int index_array[] = {index, index_args...};
            return offset_array(sizeof...(index_args) + 1, index_array);
        }

        int offset_array(const std::vector<int>& index) const;
        int offset_array(size_t size, const int* index_array) const;

        template<typename ... _Args>
        Tensor& resize(int dim_size, _Args ... dim_size_args){
            const int dim_size_array[] = {dim_size, dim_size_args...};
            return resize(sizeof...(dim_size_args) + 1, dim_size_array);
        }

        Tensor& resize(int ndims, const int* dims);
        Tensor& resize(const std::vector<int>& dims);
        Tensor& resize_single_dim(int idim, int size);
        int  count(int start_axis = 0) const;
        int device() const{return device_id_;}

        /* 把数据copy到GPU上或者copy到CPU上 */
        Tensor& to_gpu(bool copy=true);
        Tensor& to_cpu(bool copy=true);
        /* 把数据copy到GPU上或者copy到CPU上 */
        inline void* cpu() const { ((Tensor*)this)->to_cpu(); return data_->cpu(); }
        inline void* gpu() const { ((Tensor*)this)->to_gpu(); return data_->gpu(); }
        
        /* 创建模板进行泛化编程， */
        template<typename DType> inline const DType* cpu() const { return (DType*)cpu(); }
        template<typename DType> inline DType* cpu()             { return (DType*)cpu(); }
        /* 变长模板参数 ，具体可以访问 ：https://blog.csdn.net/zj510/article/details/36633603?spm=1001.2101.3001.6650.10&utm_medium=distribute.pc_relevant.none-task-blog-2%7Edefault%7EBlogCommendFromBaidu%7Edefault-10.highlightwordscore&depth_1-utm_source=distribute.pc_relevant.none-task-blog-2%7Edefault%7EBlogCommendFromBaidu%7Edefault-10.highlightwordscore  */
        /* 这里应是数据切片时使用的，下面的GPU类似 */
        template<typename DType, typename ... _Args> 
        inline DType* cpu(int i, _Args&& ... args) { return cpu<DType>() + offset(i, args...); }


        template<typename DType> inline const DType* gpu() const { return (DType*)gpu(); }
        template<typename DType> inline DType* gpu()             { return (DType*)gpu(); }

        template<typename DType, typename ... _Args> 
        inline DType* gpu(int i, _Args&& ... args) { return gpu<DType>() + offset(i, args...); }

        template<typename DType, typename ... _Args> 
        inline DType& at(int i, _Args&& ... args) { return *(cpu<DType>() + offset(i, args...)); }
        
        /* 获取数据和空间 */
        std::shared_ptr<MixMemory> get_data()             const {return data_;}
        std::shared_ptr<MixMemory> get_workspace()        const {return workspace_;}
        Tensor& set_workspace(std::shared_ptr<MixMemory> workspace) {workspace_ = workspace; return *this;}
        /* 获取流和设置流 */
        hipStream_t get_stream() const{return stream_;}
        Tensor& set_stream(hipStream_t stream){stream_ = stream; return *this;}

        Tensor& set_mat     (int n, const cv::Mat& image);
        Tensor& set_norm_mat(int n, const cv::Mat& image, float mean[3], float std[3]);
        /* 参数cpu<float>(n, c)，使用了可变长参数的功能 ，这里是获取一段数据，这里需要深挖，先放放 */
        cv::Mat at_mat(int n = 0, int c = 0) { return cv::Mat(height(), width(), CV_32F, cpu<float>(n, c)); }

        /* 设置流为异步执行 */
        Tensor& synchronize();
        const char* shape_string() const{return shape_string_;}
        const char* descriptor() const;

        /* 这部分很复杂，需要多理解 */
        Tensor& copy_from_gpu(size_t offset, const void* src, size_t num_element, int device_id = CURRENT_DEVICE_ID);

        /**
        
        # 以下代码是python中加载Tensor
        import numpy as np

        def load_tensor(file):
            
            with open(file, "rb") as f:
                binary_data = f.read()

            magic_number, ndims, dtype = np.frombuffer(binary_data, np.uint32, count=3, offset=0)
            assert magic_number == 0xFCCFE2E2, f"{file} not a tensor file."
            
            dims = np.frombuffer(binary_data, np.uint32, count=ndims, offset=3 * 4)

            if dtype == 0:
                np_dtype = np.float32
            elif dtype == 1:
                np_dtype = np.float16
            else:
                assert False, f"Unsupport dtype = {dtype}, can not convert to numpy dtype"
                
            return np.frombuffer(binary_data, np_dtype, offset=(ndims + 3) * 4).reshape(*dims)

         **/
        bool save_to_file(const std::string& file) const;

    private:
        Tensor& compute_shape_string();
        Tensor& adajust_memory_by_update_dims_or_type();
        void setup_data(std::shared_ptr<MixMemory> data);

    private:
        /* tensor的shape */
        std::vector<int> shape_;
        /* tensor所占的空间大小 */
        size_t bytes_    = 0;
        /* 数据头 包含三部分，初始化、CPU、GPU */
        DataHead head_   = DataHead::Init;
        /* 创建流的声明 */
        hipStream_t stream_ = nullptr;
        int device_id_   = 0;
        char shape_string_[100];
        char descriptor_string_[100];
        /* MixMemory获取内存或者显存 */
        std::shared_ptr<MixMemory> data_;
        std::shared_ptr<MixMemory> workspace_;
    };

    Tensor::Tensor(int n, int c, int h, int w, shared_ptr<MixMemory> data, int device_id) {
        this->device_id_ = get_device(device_id);
        descriptor_string_[0] = 0;
        setup_data(data);
        resize(n, c, h, w);
    }

    Tensor::Tensor(const std::vector<int>& dims, shared_ptr<MixMemory> data, int device_id){
        this->device_id_ = get_device(device_id);
        descriptor_string_[0] = 0;
        setup_data(data);
        resize(dims);
    }

    Tensor::Tensor(int ndims, const int* dims, shared_ptr<MixMemory> data, int device_id) {
        this->device_id_ = get_device(device_id);
        descriptor_string_[0] = 0;
        setup_data(data);
        resize(ndims, dims);
    }

    Tensor::Tensor(shared_ptr<MixMemory> data, int device_id){
        shape_string_[0] = 0;
        descriptor_string_[0] = 0;
        this->device_id_ = get_device(device_id);
        setup_data(data);
    }

    Tensor::~Tensor() {
        release();
    }

    const char* Tensor::descriptor() const{
        
        char* descriptor_ptr = (char*)descriptor_string_;
        int device_id = device();
        snprintf(descriptor_ptr, sizeof(descriptor_string_), 
            "Tensor:%p, %s, CUDA:%d", 
            data_.get(),
            shape_string_, 
            device_id
        );
        return descriptor_ptr;
    }

    Tensor& Tensor::compute_shape_string(){

        // clean string
        shape_string_[0] = 0;

        char* buffer = shape_string_;
        size_t buffer_size = sizeof(shape_string_);
        for(int i = 0; i < shape_.size(); ++i){

            int size = 0;
            if(i < shape_.size() - 1)
                size = snprintf(buffer, buffer_size, "%d x ", shape_[i]);
            else
                size = snprintf(buffer, buffer_size, "%d", shape_[i]);

            buffer += size;
            buffer_size -= size;
        }
        return *this;
    }

    /* 这里把cpu内存和GPU内存分配放到一起 */
    void Tensor::setup_data(shared_ptr<MixMemory> data){
        
        data_ = data;
        if(data_ == nullptr){
            data_ = make_shared<MixMemory>(device_id_);
        }else{
            device_id_ = data_->device_id();
        }

        head_ = DataHead::Init;
        if(data_->cpu()){
            head_ = DataHead::Host;
        }

        if(data_->gpu()){
            head_ = DataHead::Device;
        }
    }

    Tensor& Tensor::copy_from_gpu(size_t offset, const void* src, size_t num_element, int device_id){

        if(head_ == DataHead::Init)
            to_gpu(false);

        size_t offset_location = offset * element_size();
        if(offset_location >= bytes_){
            INFOE("Offset location[%lld] >= bytes_[%lld], out of range", offset_location, bytes_);
            return *this;
        }

        size_t copyed_bytes = num_element * element_size();
        size_t remain_bytes = bytes_ - offset_location;
        if(copyed_bytes > remain_bytes){
            INFOE("Copyed bytes[%lld] > remain bytes[%lld], out of range", copyed_bytes, remain_bytes);
            return *this;
        }
        
        if(head_ == DataHead::Device){
            int current_device_id = get_device(device_id);
            int gpu_device_id = device();
            if(current_device_id != gpu_device_id){
                checkCudaRuntime(hipMemcpyPeerAsync(gpu<unsigned char>() + offset_location, gpu_device_id, src, current_device_id, copyed_bytes, stream_));
                //checkCudaRuntime(hipMemcpyAsync(gpu<unsigned char>() + offset_location, src, copyed_bytes, hipMemcpyDeviceToDevice, stream_));
            }
            else{
                checkCudaRuntime(hipMemcpyAsync(gpu<unsigned char>() + offset_location, src, copyed_bytes, hipMemcpyDeviceToDevice, stream_));
            }
        }else if(head_ == DataHead::Host){
            AutoDevice auto_device_exchange(this->device());
            checkCudaRuntime(hipMemcpyAsync(cpu<unsigned char>() + offset_location, src, copyed_bytes, hipMemcpyDeviceToHost, stream_));
        }else{
            INFOE("Unsupport head type %d", head_);
        }
        return *this;
    }

    Tensor& Tensor::release() {
        data_->release_all();
        shape_.clear();
        bytes_ = 0;
        head_ = DataHead::Init;
        return *this;
    }

    bool Tensor::empty() const{
        return data_->cpu() == nullptr && data_->gpu() == nullptr;
    }

    int Tensor::count(int start_axis) const {

        if(start_axis >= 0 && start_axis < shape_.size()){
            int size = 1;
            for (int i = start_axis; i < shape_.size(); ++i) 
                size *= shape_[i];
            return size;
        }else{
            return 0;
        }
    }

    Tensor& Tensor::resize(const std::vector<int>& dims) {
        return resize(dims.size(), dims.data());
    }

    int Tensor::numel() const{
        int value = shape_.empty() ? 0 : 1;
        for(int i = 0; i < shape_.size(); ++i){
            value *= shape_[i];
        }
        return value;
    }

    Tensor& Tensor::resize_single_dim(int idim, int size){

        Assert(idim >= 0 && idim < shape_.size());

        auto new_shape = shape_;
        new_shape[idim] = size;
        return resize(new_shape);
    }

    Tensor& Tensor::resize(int ndims, const int* dims) {

        vector<int> setup_dims(ndims);
        for(int i = 0; i < ndims; ++i){
            int dim = dims[i];
            if(dim == -1){
                Assert(ndims == shape_.size());
                dim = shape_[i];
            }
            setup_dims[i] = dim;
        }
        this->shape_ = setup_dims;
        this->adajust_memory_by_update_dims_or_type();
        this->compute_shape_string();
        return *this;
    }

    Tensor& Tensor::adajust_memory_by_update_dims_or_type(){
        
        int needed_size = this->numel() * element_size();
        if(needed_size > this->bytes_){
            head_ = DataHead::Init;
        }
        this->bytes_ = needed_size;
        return *this;
    }

    Tensor& Tensor::synchronize(){ 
        AutoDevice auto_device_exchange(this->device());
        checkCudaRuntime(hipStreamSynchronize(stream_));
        return *this;
    }


    /* 
        先开辟需要大小的gpu空间，然后初始化为0 ，然后把cpu的数据转换为GPU    
    */
    Tensor& Tensor::to_gpu(bool copy) {
        /* 如果已经是GPU的数据，直接返回即可 */
        if (head_ == DataHead::Device)
            return *this;
        /* 先更新数据头信息为GPU,然后开辟GPU空间，初始化为0 */
        head_ = DataHead::Device;
        data_->gpu(bytes_);
        /* 在确定数据不为空的情况下，把数据copy到GPU上，cpu同理 */
        if (copy && data_->cpu() != nullptr) {
            AutoDevice auto_device_exchange(this->device());
            checkCudaRuntime(hipMemcpyAsync(data_->gpu(), data_->cpu(), bytes_, hipMemcpyHostToDevice, stream_));
        }
        return *this;
    }
    
    Tensor& Tensor::to_cpu(bool copy) {

        if (head_ == DataHead::Host)
            return *this;

        head_ = DataHead::Host;
        data_->cpu(bytes_);

        if (copy && data_->gpu() != nullptr) {
            AutoDevice auto_device_exchange(this->device());
            checkCudaRuntime(hipMemcpyAsync(data_->cpu(), data_->gpu(), bytes_, hipMemcpyDeviceToHost, stream_));
            checkCudaRuntime(hipStreamSynchronize(stream_));
        }
        return *this;
    }

    int Tensor::offset_array(size_t size, const int* index_array) const{

        Assert(size <= shape_.size());
        int value = 0;
        for(int i = 0; i < shape_.size(); ++i){

            if(i < size)
                value += index_array[i];

            if(i + 1 < shape_.size())
                value *= shape_[i+1];
        }
        return value;
    }

    int Tensor::offset_array(const std::vector<int>& index_array) const{
        return offset_array(index_array.size(), index_array.data());
    }

    bool Tensor::save_to_file(const std::string& file) const{

        if(empty()) return false;

        FILE* f = fopen(file.c_str(), "wb");
        if(f == nullptr) return false;

        int ndims = this->ndims();
        int dtype_ = 0;
        unsigned int head[3] = {0xFCCFE2E2, ndims, static_cast<unsigned int>(dtype_)};
        fwrite(head, 1, sizeof(head), f);
        fwrite(shape_.data(), 1, sizeof(shape_[0]) * shape_.size(), f);
        fwrite(cpu(), 1, bytes_, f);
        fclose(f);
        return true;
    }

    /////////////////////////////////class TRTInferImpl////////////////////////////////////////////////
    class Logger : public ILogger {
    public:
        virtual void log(Severity severity, const char* msg) noexcept override {

            if (severity == Severity::kINTERNAL_ERROR) {
                INFOE("NVInfer INTERNAL_ERROR: %s", msg);
                abort();
            }else if (severity == Severity::kERROR) {
                INFOE("NVInfer: %s", msg);
            }
            else  if (severity == Severity::kWARNING) {
                INFOW("NVInfer: %s", msg);
            }
            else  if (severity == Severity::kINFO) {
                INFOD("NVInfer: %s", msg);
            }
            else {
                INFOD("%s", msg);
            }
        }
    };
    static Logger gLogger;

    template<typename _T>
    static void destroy_nvidia_pointer(_T* ptr) {
        if (ptr) ptr->destroy();
    }

    /* 这个类就是构建模型的，和tensorrt官方的教程差不多， 只是这里不使用默认流，使用创建的流进行执行 */
    class EngineContext {
    public:
        virtual ~EngineContext() { destroy(); }

        void set_stream(hipStream_t stream){

            if(owner_stream_){
                if (stream_) {hipStreamDestroy(stream_);}
                owner_stream_ = false;
            }
            stream_ = stream;
        }

        bool build_model(const void* pdata, size_t size) {
            destroy();

            if(pdata == nullptr || size == 0)
                return false;

            owner_stream_ = true;
            /* 创建流 */
            checkCudaRuntime(hipStreamCreate(&stream_));
            if(stream_ == nullptr)
                return false;
            /* 下面就是标准的tensorrt的反序列化流程，不懂的可以看看官网的教程即可
               其中runtime_，engine_，context_都是类的内置变量，
            */
            runtime_ = shared_ptr<IRuntime>(createInferRuntime(gLogger), destroy_nvidia_pointer<IRuntime>);
            if (runtime_ == nullptr)
                return false;

            engine_ = shared_ptr<ICudaEngine>(runtime_->deserializeCudaEngine(pdata, size, nullptr), destroy_nvidia_pointer<ICudaEngine>);
            if (engine_ == nullptr)
                return false;

            //runtime_->setDLACore(0);
            context_ = shared_ptr<IExecutionContext>(engine_->createExecutionContext(), destroy_nvidia_pointer<IExecutionContext>);
            return context_ != nullptr;
        }

    private:
        void destroy() {
            context_.reset();
            engine_.reset();
            runtime_.reset();

            if(owner_stream_){
                if (stream_) {hipStreamDestroy(stream_);}
            }
            stream_ = nullptr;
        }

    public:
        hipStream_t stream_ = nullptr;
        bool owner_stream_ = false;
        shared_ptr<IExecutionContext> context_;
        shared_ptr<ICudaEngine> engine_;
        shared_ptr<IRuntime> runtime_ = nullptr;
    };

    /* 这里不知道大家是否存在一个疑问就是为什么很多类名称后面都有Impl结尾，这里送大家一个单词implementation，意思是实施 执行
        通常存在多态的情况下，抽象类的具体实现，即这个类是具体实现的意思
    */
    class TRTInferImpl{
    public:
        virtual ~TRTInferImpl();
        bool load(const std::string& file);
        bool load_from_memory(const void* pdata, size_t size);
        void destroy();
        void forward(bool sync);
        int get_max_batch_size();
        hipStream_t get_stream();
        void set_stream(hipStream_t stream);
        void synchronize();
        size_t get_device_memory_size();
        std::shared_ptr<MixMemory> get_workspace();
        std::shared_ptr<Tensor> input(int index = 0);
        std::string get_input_name(int index = 0);
        std::shared_ptr<Tensor> output(int index = 0);
        std::string get_output_name(int index = 0);
        std::shared_ptr<Tensor> tensor(const std::string& name);
        bool is_output_name(const std::string& name);
        bool is_input_name(const std::string& name);
        void set_input (int index, std::shared_ptr<Tensor> tensor);
        void set_output(int index, std::shared_ptr<Tensor> tensor);
        std::shared_ptr<std::vector<uint8_t>> serial_engine();

        void print();

        int num_output();
        int num_input();
        int device();

    private:
        void build_engine_input_and_outputs_mapper();

    private:
        std::vector<std::shared_ptr<Tensor>> inputs_;
        std::vector<std::shared_ptr<Tensor>> outputs_;
        std::vector<int> inputs_map_to_ordered_index_;
        std::vector<int> outputs_map_to_ordered_index_;
        std::vector<std::string> inputs_name_;
        std::vector<std::string> outputs_name_;
        std::vector<std::shared_ptr<Tensor>> orderdBlobs_;
        std::map<std::string, int> blobsNameMapper_;
        std::shared_ptr<EngineContext> context_;
        std::vector<void*> bindingsPtr_;
        std::shared_ptr<MixMemory> workspace_;
        int device_ = 0;
    };

    ////////////////////////////////////////////////////////////////////////////////////
    TRTInferImpl::~TRTInferImpl(){
        destroy();
    }

    void TRTInferImpl::destroy() {

        int old_device = 0;
        checkCudaRuntime(hipGetDevice(&old_device));
        checkCudaRuntime(hipSetDevice(device_));
        this->context_.reset();
        this->blobsNameMapper_.clear();
        this->outputs_.clear();
        this->inputs_.clear();
        this->inputs_name_.clear();
        this->outputs_name_.clear();
        checkCudaRuntime(hipSetDevice(old_device));
    }

    void TRTInferImpl::print(){
        if(!context_){
            INFOW("Infer print, nullptr.");
            return;
        }

        INFO("Infer %p detail", this);
        INFO("\tMax Batch Size: %d", this->get_max_batch_size());
        INFO("\tInputs: %d", inputs_.size());
        for(int i = 0; i < inputs_.size(); ++i){
            auto& tensor = inputs_[i];
            auto& name = inputs_name_[i];
            INFO("\t\t%d.%s : shape {%s}", i, name.c_str(), tensor->shape_string());
        }

        INFO("\tOutputs: %d", outputs_.size());
        for(int i = 0; i < outputs_.size(); ++i){
            auto& tensor = outputs_[i];
            auto& name = outputs_name_[i];
            INFO("\t\t%d.%s : shape {%s}", i, name.c_str(), tensor->shape_string());
        }
    }

    std::shared_ptr<std::vector<uint8_t>> TRTInferImpl::serial_engine() {
        auto memory = this->context_->engine_->serialize();
        auto output = make_shared<std::vector<uint8_t>>((uint8_t*)memory->data(), (uint8_t*)memory->data()+memory->size());
        memory->destroy();
        return output;
    }

    bool TRTInferImpl::load_from_memory(const void* pdata, size_t size) {

        if (pdata == nullptr || size == 0)
            return false;

        context_.reset(new EngineContext());

        //build model
        if (!context_->build_model(pdata, size)) {
            context_.reset();
            return false;
        }

        workspace_.reset(new MixMemory());
        hipGetDevice(&device_);
        build_engine_input_and_outputs_mapper();
        return true;
    }

    static std::vector<uint8_t> load_file(const string& file){

        ifstream in(file, ios::in | ios::binary);
        if (!in.is_open())
            return {};

        in.seekg(0, ios::end);
        size_t length = in.tellg();

        std::vector<uint8_t> data;
        if (length > 0){
            in.seekg(0, ios::beg);
            data.resize(length);

            in.read((char*)&data[0], length);
        }
        in.close();
        return data;
    }

    bool TRTInferImpl::load(const std::string& file) {
        /* 反序列化数据 */
        auto data = load_file(file);
        if (data.empty())
            return false;

        context_.reset(new EngineContext());

        //build model
        if (!context_->build_model(data.data(), data.size())) {
            context_.reset();
            return false;
        }

        workspace_.reset(new MixMemory());
        hipGetDevice(&device_);
        /* 输入输出结果绑定或者是映射  */
        build_engine_input_and_outputs_mapper();
        return true;
    }

    size_t TRTInferImpl::get_device_memory_size() {
        EngineContext* context = (EngineContext*)this->context_.get();
        return context->context_->getEngine().getDeviceMemorySize();
    }

    void TRTInferImpl::build_engine_input_and_outputs_mapper() {
        /*  引擎执行上下文 */
        EngineContext* context = (EngineContext*)this->context_.get();
        /*  获取输入输出的个数  */
        int nbBindings = context->engine_->getNbBindings();
        /*  获取最大的batch  */
        int max_batchsize = context->engine_->getMaxBatchSize();

        inputs_.clear();
        inputs_name_.clear();
        outputs_.clear();
        outputs_name_.clear();
        orderdBlobs_.clear();
        bindingsPtr_.clear();
        blobsNameMapper_.clear();
        for (int i = 0; i < nbBindings; ++i) {
            /* 获取维度dims = {nbDims=4 d=0x000000c1e77ff2bc {-1, 3, 640, 640, 0, 0, 0, 0} }， dims = {nbDims=3 d=0x000000c1e77ff2bc {-1, 25200, 85, 0, 0, 0, 0, 0} } */
            auto dims = context->engine_->getBindingDimensions(i);
            /* 获取数据类型 */
            auto type = context->engine_->getBindingDataType(i);
            /* 获取绑定的名称 */
            const char* bindingName = context->engine_->getBindingName(i);
            /* 设置批次 dims = {nbDims=4 d=0x000000c1e77ff2bc {16, 3, 640, 640, 0, 0, 0, 0} } ， dims = {nbDims=3 d=0x000000c1e77ff2bc {16, 25200, 85, 0, 0, 0, 0, 0} } */
            dims.d[0] = max_batchsize;
            /* 创建tensor   */
            auto newTensor = make_shared<Tensor>(dims.nbDims, dims.d);
            /* 把模型的流和tensor绑定 */
            newTensor->set_stream(this->context_->stream_);
            /* 给tensor开辟空间 */
            newTensor->set_workspace(this->workspace_);
            /* 判断是输入还是输出 */
            if (context->engine_->bindingIsInput(i)) {
                //if is input
                inputs_.push_back(newTensor);
                inputs_name_.push_back(bindingName);
                inputs_map_to_ordered_index_.push_back(orderdBlobs_.size());
            }
            else {
                //if is output
                outputs_.push_back(newTensor);
                outputs_name_.push_back(bindingName);
                outputs_map_to_ordered_index_.push_back(orderdBlobs_.size());
            }
            blobsNameMapper_[bindingName] = i;
            orderdBlobs_.push_back(newTensor);
        }
        bindingsPtr_.resize(orderdBlobs_.size());
    }

    void TRTInferImpl::set_stream(hipStream_t stream){
        this->context_->set_stream(stream);

        for(auto& t : orderdBlobs_)
            t->set_stream(stream);
    }

    hipStream_t TRTInferImpl::get_stream() {
        return this->context_->stream_;
    }

    int TRTInferImpl::device() {
        return device_;
    }

    void TRTInferImpl::synchronize() {
        checkCudaRuntime(hipStreamSynchronize(context_->stream_));
    }

    bool TRTInferImpl::is_output_name(const std::string& name){
        return std::find(outputs_name_.begin(), outputs_name_.end(), name) != outputs_name_.end();
    }

    bool TRTInferImpl::is_input_name(const std::string& name){
        return std::find(inputs_name_.begin(), inputs_name_.end(), name) != inputs_name_.end();
    }

    void TRTInferImpl::forward(bool sync) {

        EngineContext* context = (EngineContext*)context_.get();
        int inputBatchSize = inputs_[0]->size(0);
        for(int i = 0; i < context->engine_->getNbBindings(); ++i){
            auto dims = context->engine_->getBindingDimensions(i);
            auto type = context->engine_->getBindingDataType(i);
            dims.d[0] = inputBatchSize;
            if(context->engine_->bindingIsInput(i)){
                context->context_->setBindingDimensions(i, dims);
            }
        }

        for (int i = 0; i < outputs_.size(); ++i) {
            outputs_[i]->resize_single_dim(0, inputBatchSize);
            outputs_[i]->to_gpu(false);
        }

        for (int i = 0; i < orderdBlobs_.size(); ++i)
            bindingsPtr_[i] = orderdBlobs_[i]->gpu();

        void** bindingsptr = bindingsPtr_.data();
        //bool execute_result = context->context_->enqueue(inputBatchSize, bindingsptr, context->stream_, nullptr);
        bool execute_result = context->context_->enqueueV2(bindingsptr, context->stream_, nullptr);
        if(!execute_result){
            auto code = hipGetLastError();
            INFOF("execute fail, code %d[%s], message %s", code, hipGetErrorName(code), hipGetErrorString(code));
        }

        if (sync) {
            synchronize();
        }
    }

    std::shared_ptr<MixMemory> TRTInferImpl::get_workspace() {
        return workspace_;
    }

    int TRTInferImpl::num_input() {
        return this->inputs_.size();
    }

    int TRTInferImpl::num_output() {
        return this->outputs_.size();
    }

    void TRTInferImpl::set_input (int index, std::shared_ptr<Tensor> tensor){
        Assert(index >= 0 && index < inputs_.size());
        this->inputs_[index] = tensor;

        int order_index = inputs_map_to_ordered_index_[index];
        this->orderdBlobs_[order_index] = tensor;
    }

    void TRTInferImpl::set_output(int index, std::shared_ptr<Tensor> tensor){
        Assert(index >= 0 && index < outputs_.size());
        this->outputs_[index] = tensor;

        int order_index = outputs_map_to_ordered_index_[index];
        this->orderdBlobs_[order_index] = tensor;
    }

    std::shared_ptr<Tensor> TRTInferImpl::input(int index) {
        Assert(index >= 0 && index < inputs_name_.size());
        return this->inputs_[index];
    }

    std::string TRTInferImpl::get_input_name(int index){
        Assert(index >= 0 && index < inputs_name_.size());
        return inputs_name_[index];
    }

    std::shared_ptr<Tensor> TRTInferImpl::output(int index) {
        Assert(index >= 0 && index < outputs_.size());
        return outputs_[index];
    }

    std::string TRTInferImpl::get_output_name(int index){
        Assert(index >= 0 && index < outputs_name_.size());
        return outputs_name_[index];
    }

    int TRTInferImpl::get_max_batch_size() {
        Assert(this->context_ != nullptr);
        return this->context_->engine_->getMaxBatchSize();
    }

    std::shared_ptr<Tensor> TRTInferImpl::tensor(const std::string& name) {
        Assert(this->blobsNameMapper_.find(name) != this->blobsNameMapper_.end());
        return orderdBlobs_[blobsNameMapper_[name]];
    }

    std::shared_ptr<TRTInferImpl> load_infer(const string& file) {
        /* 实例化一个推理对象 */
        std::shared_ptr<TRTInferImpl> infer(new TRTInferImpl());
        /* 加载trt文件，并反序列化，这里包含了模型的输入输出的绑定和流的设定 */
        if (!infer->load(file))
            infer.reset();
        return infer;
    }

    //////////////////////////////class MonopolyAllocator//////////////////////////////////////
    /* 独占分配器
       通过对tensor做独占管理，具有max_batch * 2个tensor，通过query获取一个
       当推理结束后，该tensor释放使用权，即可交给下一个图像使用，内存实现复用
    */
    template<class _ItemType>
    class MonopolyAllocator{
    public:
        class MonopolyData{
        public:
            std::shared_ptr<_ItemType>& data(){ return data_; }
            void release(){manager_->release_one(this);}

        private:
            MonopolyData(MonopolyAllocator* pmanager){manager_ = pmanager;}

        private:
            friend class MonopolyAllocator;
            MonopolyAllocator* manager_ = nullptr;
            std::shared_ptr<_ItemType> data_;
            bool available_ = true;
        };
        typedef std::shared_ptr<MonopolyData> MonopolyDataPointer;

        MonopolyAllocator(int size){
            capacity_ = size;
            num_available_ = size;
            datas_.resize(size);

            for(int i = 0; i < size; ++i)
                datas_[i] = std::shared_ptr<MonopolyData>(new MonopolyData(this));
        }

        virtual ~MonopolyAllocator(){
            run_ = false;
            cv_.notify_all();
            
            std::unique_lock<std::mutex> l(lock_);
            cv_exit_.wait(l, [&](){
                return num_wait_thread_ == 0;
            });
        }

        MonopolyDataPointer query(int timeout = 10000){

            std::unique_lock<std::mutex> l(lock_);
            if(!run_) return nullptr;
            
            if(num_available_ == 0){
                num_wait_thread_++;

                auto state = cv_.wait_for(l, std::chrono::milliseconds(timeout), [&](){
                    return num_available_ > 0 || !run_;
                });

                num_wait_thread_--;
                cv_exit_.notify_one();

                // timeout, no available, exit program
                if(!state || num_available_ == 0 || !run_)
                    return nullptr;
            }

            auto item = std::find_if(datas_.begin(), datas_.end(), [](MonopolyDataPointer& item){return item->available_;});
            if(item == datas_.end())
                return nullptr;
            
            (*item)->available_ = false;
            num_available_--;
            return *item;
        }

        int num_available(){
            return num_available_;
        }

        int capacity(){
            return capacity_;
        }

    private:
        void release_one(MonopolyData* prq){
            std::unique_lock<std::mutex> l(lock_);
            if(!prq->available_){
                prq->available_ = true;
                num_available_++;
                cv_.notify_one();
            }
        }

    private:
        std::mutex lock_;
        std::condition_variable cv_;
        std::condition_variable cv_exit_;
        std::vector<MonopolyDataPointer> datas_;
        int capacity_ = 0;
        volatile int num_available_ = 0;
        volatile int num_wait_thread_ = 0;
        volatile bool run_ = true;
    };


    /////////////////////////////////////////class ThreadSafedAsyncInfer/////////////////////////////////////////////
    /* 异步线程安全的推理器
       通过异步线程启动，使得调用方允许任意线程调用把图像做输入，并通过future来获取异步结果
    */
    template<class Input, class Output, class StartParam=std::tuple<std::string, int>, class JobAdditional=int>
    class ThreadSafedAsyncInfer{
    public:
        /* 定义结构体的目的是便于接收模板传入的参数和后面的使用方便 */
        struct Job{
            Input input;                                                /* 输入相关参数 */
            Output output;                                              /* 输出相关参数 */
            JobAdditional additional;                                   /* 预处理和后处理相关矩阵 */
            MonopolyAllocator<Tensor>::MonopolyDataPointer mono_tensor; /* 独一的tensor */
            std::shared_ptr<std::promise<Output>> pro;                  /* promise,获取相关结果使用的 */
        };

        virtual ~ThreadSafedAsyncInfer(){
            stop();
        }

        void stop(){
            run_ = false;
            cond_.notify_all();

            ////////////////////////////////////////// cleanup jobs
            {
                std::unique_lock<std::mutex> l(jobs_lock_);
                while(!jobs_.empty()){
                    auto& item = jobs_.front();
                    if(item.pro)
                        item.pro->set_value(Output());
                    jobs_.pop();
                }
            };

            if(worker_){
                worker_->join();
                worker_.reset();
            }
        }

        /* 开始启动，主要功能是启动完成，等待结果 */ 
        bool startup(const StartParam& param){
            run_ = true;
            /* 这里使用的promise和future的目的只是通知模型加载和参数配置完成，等待后面的数据图片任务 */
            std::promise<bool> pro;
            start_param_ = param;
            /* 开启线程，完成初始化工作后，等待预处理完成的图片，然后进行推理工作 */
            worker_      = std::make_shared<std::thread>(&ThreadSafedAsyncInfer::worker, this, std::ref(pro));
            /* 主线程来到这里会阻塞，阻塞来源上面的promise的pro对象，需要等待pro对象的返回 */
            return pro.get_future().get();
        }

        virtual std::shared_future<Output> commit(const Input& input){

            Job job;
            job.pro = std::make_shared<std::promise<Output>>();
            if(!preprocess(job, input)){
                job.pro->set_value(Output());
                return job.pro->get_future();
            }
            
            ///////////////////////////////////////////////////////////
            {
                std::unique_lock<std::mutex> l(jobs_lock_);
                jobs_.push(job);
            };
            cond_.notify_one();
            return job.pro->get_future();
        }

        virtual std::vector<std::shared_future<Output>> commits(const std::vector<Input>& inputs){

            /* batch_size的大小 */
            int batch_size = std::min((int)inputs.size(), this->tensor_allocator_->capacity());
            /* 创建一个job的vector，因此使用的是batch进行推理即多张图片的推理 */
            std::vector<Job> jobs(inputs.size());
            /* 创建一个输出结果接收vector */
            std::vector<std::shared_future<Output>> results(inputs.size());

            int nepoch = (inputs.size() + batch_size - 1) / batch_size;
            for(int epoch = 0; epoch < nepoch; ++epoch){
                /* 输入图片 */
                int begin = epoch * batch_size;
                int end   = std::min((int)inputs.size(), begin + batch_size);
                /* 遍历图片 */
                for(int i = begin; i < end; ++i){
                    /* 实例化一个Job对象，用作数据的传输 */
                    Job& job = jobs[i];
                    /* 每一张图片都对应这个JOb的结构体，这里对promise进行实例化填充 */
                    job.pro = std::make_shared<std::promise<Output>>();
                    /* 开始进行预处理，其中job包含了所需的参数数据，到预处理进行填充或者使用 */
                    if(!preprocess(job, inputs[i])){
                        job.pro->set_value(Output());
                    }
                    /* 把图片的结果进行保存，来源这里解码的job.pro->set_value(image_based_boxes); */
                    results[i] = job.pro->get_future();
                }

                /* 上面预处理的数据还是在jobs中，因此直接把jobs的数据压入队列即可，然后唤醒工作线程*/
                {
                    /* 上锁，上锁的有效范围就是该{}内，程序离开该区域，自动解锁 */
                    std::unique_lock<std::mutex> l(jobs_lock_);
                    for(int i = begin; i < end; ++i){
                        jobs_.emplace(std::move(jobs[i]));
                    };
                }
                cond_.notify_one();
            }
            return results;
        }

    protected:
        virtual void worker(std::promise<bool>& result) = 0;
        virtual bool preprocess(Job& job, const Input& input) = 0;
        
        virtual bool get_jobs_and_wait(std::vector<Job>& fetch_jobs, int max_size){
            /* 定义一个互斥量锁，目的是当存在多线程同时获取jobs队列的数据时的安全保护机制，但是该工程只有当前线程，因此不存在竞争关系  */
            std::unique_lock<std::mutex> l(jobs_lock_);
            /*   等待唤醒  */ 
            cond_.wait(l, [&](){
                return !run_ || !jobs_.empty();
            });
            /* 经过commits的数据加载和预处理后，并把数据传入到队列jobs_中，同时唤醒子线程开始处理 */
            if(!run_) return false;
            
            /* 唤醒后开始工作 */
            fetch_jobs.clear();
            /* 把jobs_队列 里的数据填充到fetch_jobs， 供后面处理 */
            for(int i = 0; i < max_size && !jobs_.empty(); ++i){
                fetch_jobs.emplace_back(std::move(jobs_.front()));
                jobs_.pop();
            }
            return true;
        }

        virtual bool get_job_and_wait(Job& fetch_job){

            std::unique_lock<std::mutex> l(jobs_lock_);
            cond_.wait(l, [&](){
                return !run_ || !jobs_.empty();
            });

            if(!run_) return false;
            
            fetch_job = std::move(jobs_.front());
            jobs_.pop();
            return true;
        }

    protected:
        StartParam start_param_;
        std::atomic<bool> run_; /* 原子操作 */
        std::mutex jobs_lock_;
        std::queue<Job> jobs_;
        std::shared_ptr<std::thread> worker_;
        std::condition_variable cond_;
        std::shared_ptr<MonopolyAllocator<Tensor>> tensor_allocator_;
    };


    ///////////////////////////////////class YoloTRTInferImpl//////////////////////////////////////
    /* Yolo的具体实现
        通过上述类的特性，实现预处理的计算重叠、异步垮线程调用，最终拼接为多个图为一个batch进行推理。最大化的利用
        显卡性能，实现高性能高可用好用的yolo推理
    */
    const char* type_name(Type type){
        switch(type){
        case Type::V5: return "YoloV5";
        case Type::X: return "YoloX";
        default: return "Unknow";
        }
    }

    struct AffineMatrix{
        float i2d[6];       // image to dst(network), 2x3 matrix
        float d2i[6];       // dst to image, 2x3 matrix

        void compute(const cv::Size& from, const cv::Size& to){
            float scale_x = to.width / (float)from.width;
            float scale_y = to.height / (float)from.height;
            float scale = std::min(scale_x, scale_y);
            i2d[0] = scale;  i2d[1] = 0;  i2d[2] = -scale * from.width  * 0.5  + to.width * 0.5;
            i2d[3] = 0;  i2d[4] = scale;  i2d[5] = -scale * from.height * 0.5 + to.height * 0.5;

            cv::Mat m2x3_i2d(2, 3, CV_32F, i2d);
            cv::Mat m2x3_d2i(2, 3, CV_32F, d2i);
            cv::invertAffineTransform(m2x3_i2d, m2x3_d2i);
        }

        cv::Mat i2d_mat(){
            return cv::Mat(2, 3, CV_32F, i2d);
        }
    };

    using ThreadSafedAsyncInferImpl = ThreadSafedAsyncInfer
    <
        cv::Mat,                    // input
        BoxArray,                   // output
        tuple<string, int>,         // start param
        AffineMatrix                // additional
    >;
    class YoloTRTInferImpl : public Infer, public ThreadSafedAsyncInferImpl{
    public:

        /* 要求在TRTInferImpl里面执行stop，而不是在基类执行stop */
        virtual ~YoloTRTInferImpl(){
            stop();
        }

        virtual bool startup(const string& file, Type type, int gpuid, float confidence_threshold, float nms_threshold){

            if(type == Type::V5){
                /* 归一化，获取归一化的参数，这里可以设置归一化参数 */
                normalize_ = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
            }else if(type == Type::X){
                //float mean[] = {0.485, 0.456, 0.406};
                //float std[]  = {0.229, 0.224, 0.225};
                //normalize_ = Norm::mean_std(mean, std, 1/255.0f, ChannelType::Invert);
                normalize_ = Norm::None();
            }else{
                INFOE("Unsupport type %d", type);
            }
            
            confidence_threshold_ = confidence_threshold;
            nms_threshold_        = nms_threshold;
            return ThreadSafedAsyncInferImpl::startup(make_tuple(file, gpuid));
        }

        virtual void worker(promise<bool>& result) override{
            /*  解析传入的参数，分别是模型路径和调用GPUid号 */
            string file = get<0>(start_param_);
            int gpuid   = get<1>(start_param_);
            /*  设置使用GPU */
            set_device(gpuid);
            /*  加载模型反序列化，绑定cuda流,绑定输入输出等操作 */
            auto engine = load_infer(file);
            if(engine == nullptr){
                INFOE("Engine %s load failed", file.c_str());
                result.set_value(false);
                return;
            }
            /* 打印引擎相关信息 */
            engine->print();
            /* 设置bbox的最大数 */
            const int MAX_IMAGE_BBOX  = 1024;
            /* 每个bbox的携带的数据 */
            const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
            /* 定义一个仿射矩阵的tensor */
            Tensor affin_matrix_device;
            /* 定义一个输出的tensor */
            Tensor output_array_device;
            /* 获取引擎的相关信息 */
            int max_batch_size = engine->get_max_batch_size();
            auto input         = engine->tensor("images");
            auto output        = engine->tensor("output");
            int num_classes    = output->size(2) - 5;

            input_width_       = input->size(3);
            input_height_      = input->size(2);
            /* 分配GPU显存，显存的大小为max_batch_size * 2 */
            tensor_allocator_  = make_shared<MonopolyAllocator<Tensor>>(max_batch_size * 2);
            stream_            = engine->get_stream();
            gpu_               = gpuid;
            /* 执行下面的代码，会使得主线程继续执行， 在这里设置阻塞的原因，可能设计者认为初始化会慢于任务的到来 */
            result.set_value(true);

            input->resize_single_dim(0, max_batch_size).to_gpu();
            affin_matrix_device.set_stream(stream_);

            /* 这里8个值的目的是保证 8 * sizeof(float) % 32 == 0 */ 
            affin_matrix_device.resize(max_batch_size, 8).to_gpu();

            /* 这里的 1 + MAX_IMAGE_BBOX结构是，counter + bboxes ... */ 
            output_array_device.resize(max_batch_size, 1 + MAX_IMAGE_BBOX * NUM_BOX_ELEMENT).to_gpu();

            vector<Job> fetch_jobs;

            /* 上面的准备工作做完后，将等待预处理后的图片过来，进行处理 */
            while(get_jobs_and_wait(fetch_jobs, max_batch_size)){

                /* 一旦进来说明有图片数据 ，获取图片的张数 */
                int infer_batch_size = fetch_jobs.size();
                input->resize_single_dim(0, infer_batch_size);
                /* 下面从队列取出job，把对应的仿射矩阵和预处理好的图片数据送到模型的输入 */
                /* 其中input就是engine对象的方法，该方法实际上是把预处理的数据传给engine的内部属性inputs_  */
                for(int ibatch = 0; ibatch < infer_batch_size; ++ibatch){
                    auto& job  = fetch_jobs[ibatch];
                    auto& mono = job.mono_tensor->data();
                    affin_matrix_device.copy_from_gpu(affin_matrix_device.offset(ibatch), mono->get_workspace()->gpu(), 6);
                    input->copy_from_gpu(input->offset(ibatch), mono->gpu(), mono->count());
                    job.mono_tensor->release();
                }
                /* 开始推理 */
                engine->forward(false);
                output_array_device.to_gpu(false);
                /* 下面进行解码，解码后面在详细研究 */
                for(int ibatch = 0; ibatch < infer_batch_size; ++ibatch){
                    
                    auto& job                 = fetch_jobs[ibatch];/* 图片数据 */
                    float* image_based_output = output->gpu<float>(ibatch);
                    float* output_array_ptr   = output_array_device.gpu<float>(ibatch);
                    auto affine_matrix        = affin_matrix_device.gpu<float>(ibatch);
                    checkCudaRuntime(hipMemsetAsync(output_array_ptr, 0, sizeof(int), stream_));
                    decode_kernel_invoker(image_based_output, output->size(1), num_classes, confidence_threshold_, nms_threshold_, affine_matrix, output_array_ptr, MAX_IMAGE_BBOX, stream_);
                }

                output_array_device.to_cpu();
                for(int ibatch = 0; ibatch < infer_batch_size; ++ibatch){
                    float* parray = output_array_device.cpu<float>(ibatch);
                    int count     = min(MAX_IMAGE_BBOX, (int)*parray);
                    auto& job     = fetch_jobs[ibatch];
                    auto& image_based_boxes   = job.output;
                    for(int i = 0; i < count; ++i){
                        float* pbox  = parray + 1 + i * NUM_BOX_ELEMENT;
                        int label    = pbox[5];
                        int keepflag = pbox[6];
                        if(keepflag == 1){
                            image_based_boxes.emplace_back(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], label);
                        }
                    }
                    job.pro->set_value(image_based_boxes);
                }
                fetch_jobs.clear();
            }
            stream_ = nullptr;
            tensor_allocator_.reset();
            INFO("Engine destroy.");
        }

        virtual bool preprocess(Job& job, const Mat& image) override{

            if(tensor_allocator_ == nullptr){
                INFOE("tensor_allocator_ is nullptr");
                return false;
            }

            job.mono_tensor = tensor_allocator_->query();
            if(job.mono_tensor == nullptr){
                INFOE("Tensor allocator query failed.");
                return false;
            }
            
            /* 配置gpu */
            AutoDevice auto_device(gpu_);
            /* 获取job里面的tensor的数据地址，第一次为nullptr */
            /* 这里需要理解的不是创建了新的tensor对象，只是把job的tensor地址拿出来使用，数据还是job指定的 */
            auto& tensor = job.mono_tensor->data(); 
            if(tensor == nullptr){
                // not init
                tensor = make_shared<Tensor>();
                tensor->set_workspace(make_shared<MixMemory>());
            }
            /* 获取输入模型的shape， input_width_和input_height_在模型创建时从模型获取 */
            Size input_size(input_width_, input_height_);
            /* 把当前的图片大小和模型所需的大小，输入进去获取仿射变换的矩阵 */
            job.additional.compute(image.size(), input_size);
            /* 把tensor和流绑定，后续都会使用这个流进行处理，流的创建也是在模型创建时创建 */
            tensor->set_stream(stream_);
            /* 把tensor  resize一下，此时的tensor还未填充数据 */
            tensor->resize(1, 3, input_height_, input_width_);

            /* GPU的显存设置 主要考虑的是仿射矩阵和图片数据的传输，这里需要深入理为什么这样做？ */
            /* 获取图片的大小 */
            size_t size_image      = image.cols * image.rows * 3;
            /* 获取仿射矩阵的大小，同时进行字节对齐 */
            size_t size_matrix     = upbound(sizeof(job.additional.d2i), 32);
            /* 获取创建内存的对象 */
            auto workspace         = tensor->get_workspace();
            /* 创建GPU显存，并返回起始地址，同时获取的空间大小是图片和仿射矩阵一起的大小 */
            uint8_t* gpu_workspace        = (uint8_t*)workspace->gpu(size_matrix + size_image);
            /* 这里显存填充数据是通过先填充仿射矩阵的，在填充图片的数据，那么起始位置应该是仿射矩阵的地址，因此如下 */
            float*   affine_matrix_device = (float*)gpu_workspace;
            /* 显存起始地址加上仿射矩阵地址就是图片的地址，因此如下，下面的cpu的类似 */
            uint8_t* image_device         = size_matrix + gpu_workspace;

            uint8_t* cpu_workspace        = (uint8_t*)workspace->cpu(size_matrix + size_image);
            float* affine_matrix_host     = (float*)cpu_workspace;
            uint8_t* image_host           = size_matrix + cpu_workspace;

            //checkCudaRuntime(hipMemcpyAsync(image_host,   image.data, size_image, hipMemcpyHostToHost,   stream_));
            // speed up
            /* 具体的拷贝上述说明相同 */
            memcpy(image_host, image.data, size_image);
            memcpy(affine_matrix_host, job.additional.d2i, sizeof(job.additional.d2i));
            checkCudaRuntime(hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream_));
            checkCudaRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(job.additional.d2i), hipMemcpyHostToDevice, stream_));

            /* 这里将开始进行仿射变换其中输入的主要是image_device和affine_matrix_device， 输出主要是tensor->gpu<float>() */
            warp_affine_bilinear_and_normalize_plane(
                image_device,         image.cols * 3,       image.cols,       image.rows, 
                tensor->gpu<float>(), input_width_,         input_height_, 
                affine_matrix_device, 114, 
                normalize_, stream_
            );
            /* 这里还需要说明一下 tensor的最终地址还是job里的地址，只是这块地址是固定的，两个batch的大小，因此这里处理完就结束了，但是
                数据已经在job里了 
                inline void* gpu() const { ((Tensor*)this)->to_gpu(); return data_->gpu(); }
            */
            return true;
        }

        virtual vector<shared_future<BoxArray>> commits(const vector<Mat>& images) override{
            return ThreadSafedAsyncInferImpl::commits(images);
        }

        virtual std::shared_future<BoxArray> commit(const Mat& image) override{
            return ThreadSafedAsyncInferImpl::commit(image);
        }

    private:
        int input_width_            = 0;
        int input_height_           = 0;
        int gpu_                    = 0;
        float confidence_threshold_ = 0;
        float nms_threshold_        = 0;
        hipStream_t stream_       = nullptr;
        Norm normalize_;
    };

    void image_to_tensor(const cv::Mat& image, shared_ptr<Tensor>& tensor, Type type, int ibatch){

        Norm normalize;
        if(type == Type::V5){
            normalize = Norm::alpha_beta(1 / 255.0f, 0.0f, ChannelType::SwapRB);
        }else if(type == Type::X){
            //float mean[] = {0.485, 0.456, 0.406};
            //float std[]  = {0.229, 0.224, 0.225};
            //normalize_ = CUDAKernel::Norm::mean_std(mean, std, 1/255.0f, CUDAKernel::ChannelType::Invert);
            normalize = Norm::None();
        }else{
            INFOE("Unsupport type %d", type);
        }
        
        Size input_size(tensor->size(3), tensor->size(2));
        AffineMatrix affine;
        affine.compute(image.size(), input_size);

        size_t size_image      = image.cols * image.rows * 3;
        size_t size_matrix     = upbound(sizeof(affine.d2i), 32);
        auto workspace         = tensor->get_workspace();
        uint8_t* gpu_workspace        = (uint8_t*)workspace->gpu(size_matrix + size_image);
        float*   affine_matrix_device = (float*)gpu_workspace;
        uint8_t* image_device         = size_matrix + gpu_workspace;

        uint8_t* cpu_workspace        = (uint8_t*)workspace->cpu(size_matrix + size_image);
        float* affine_matrix_host     = (float*)cpu_workspace;
        uint8_t* image_host           = size_matrix + cpu_workspace;
        auto stream                   = tensor->get_stream();

        memcpy(image_host, image.data, size_image);
        memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
        checkCudaRuntime(hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream));
        checkCudaRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(affine.d2i), hipMemcpyHostToDevice, stream));

        warp_affine_bilinear_and_normalize_plane(
            image_device,               image.cols * 3,       image.cols,       image.rows, 
            tensor->gpu<float>(ibatch), input_size.width,     input_size.height, 
            affine_matrix_device, 114, 
            normalize, stream
        );
    }

    shared_ptr<Infer> create_infer(const string& engine_file, Type type, int gpuid, float confidence_threshold, float nms_threshold){
        /* 创建一个推理实例，该实例具备了引擎的创建、加载模型，反序列化，创建线程等一系列操作， */
        shared_ptr<YoloTRTInferImpl> instance(new YoloTRTInferImpl());
        if(!instance->startup(engine_file, type, gpuid, confidence_threshold, nms_threshold)){
            instance.reset();
        }
        return instance;
    }

    //////////////////////////////////////Compile Model/////////////////////////////////////////////////////////////

    const char* mode_string(Mode type) {
        switch (type) {
        case Mode::FP32:
            return "FP32";
        case Mode::FP16:
            return "FP16";
        case Mode::INT8:
            return "INT8";
        default:
            return "UnknowCompileMode";
        }
    }

    typedef std::function<void(int current, int count, const std::vector<std::string>& files, std::shared_ptr<Tensor>& tensor)> Int8Process;

    class Int8EntropyCalibrator : public IInt8EntropyCalibrator2{
    public:
        Int8EntropyCalibrator(const vector<string>& imagefiles, nvinfer1::Dims dims, const Int8Process& preprocess) {

            Assert(preprocess != nullptr);
            this->dims_ = dims;
            this->allimgs_ = imagefiles;
            this->preprocess_ = preprocess;
            this->fromCalibratorData_ = false;
            files_.resize(dims.d[0]);
            checkCudaRuntime(hipStreamCreate(&stream_));
        }

        Int8EntropyCalibrator(const vector<uint8_t>& entropyCalibratorData, nvinfer1::Dims dims, const Int8Process& preprocess) {
            Assert(preprocess != nullptr);

            this->dims_ = dims;
            this->entropyCalibratorData_ = entropyCalibratorData;
            this->preprocess_ = preprocess;
            this->fromCalibratorData_ = true;
            files_.resize(dims.d[0]);
            checkCudaRuntime(hipStreamCreate(&stream_));
        }

        virtual ~Int8EntropyCalibrator(){
            checkCudaRuntime(hipStreamDestroy(stream_));
        }

        int getBatchSize() const noexcept {
            return dims_.d[0];
        }

        bool next() {
            int batch_size = dims_.d[0];
            if (cursor_ + batch_size > allimgs_.size())
                return false;

            int old_cursor = cursor_;
            for(int i = 0; i < batch_size; ++i)
                files_[i] = allimgs_[cursor_++];

            if (!tensor_){
                tensor_.reset(new Tensor(dims_.nbDims, dims_.d));
                tensor_->set_stream(stream_);
                tensor_->set_workspace(make_shared<MixMemory>());
            }

            preprocess_(old_cursor, allimgs_.size(), files_, tensor_);
            return true;
        }

        bool getBatch(void* bindings[], const char* names[], int nbBindings) noexcept {
            if (!next()) return false;
            bindings[0] = tensor_->gpu();
            return true;
        }

        const vector<uint8_t>& getEntropyCalibratorData() {
            return entropyCalibratorData_;
        }

        const void* readCalibrationCache(size_t& length) noexcept {
            if (fromCalibratorData_) {
                length = this->entropyCalibratorData_.size();
                return this->entropyCalibratorData_.data();
            }

            length = 0;
            return nullptr;
        }

        virtual void writeCalibrationCache(const void* cache, size_t length) noexcept {
            entropyCalibratorData_.assign((uint8_t*)cache, (uint8_t*)cache + length);
        }

    private:
        Int8Process preprocess_;
        vector<string> allimgs_;
        size_t batchCudaSize_ = 0;
        int cursor_ = 0;
        nvinfer1::Dims dims_;
        vector<string> files_;
        shared_ptr<Tensor> tensor_;
        vector<uint8_t> entropyCalibratorData_;
        bool fromCalibratorData_ = false;
        hipStream_t stream_ = nullptr;
    };

    bool compile(
        Mode mode, Type type,
        unsigned int max_batch_size,
        const string& source_onnx,
        const string& saveto,
        size_t max_workspace_size,
        const std::string& int8_images_folder,
        const std::string& int8_entropy_calibrator_cache_file) {

        bool hasEntropyCalibrator = false;
        vector<uint8_t> entropyCalibratorData;
        vector<string> entropyCalibratorFiles;

        auto int8process = [=](int current, int count, const vector<string>& files, shared_ptr<Tensor>& tensor){

            for(int i = 0; i < files.size(); ++i){

                auto& file = files[i];
                INFO("Int8 load %d / %d, %s", current + i + 1, count, file.c_str());

                auto image = cv::imread(file);
                if(image.empty()){
                    INFOE("Load image failed, %s", file.c_str());
                    continue;
                }
                image_to_tensor(image, tensor, type, i);
            }
            tensor->synchronize();
        };

        if (mode == Mode::INT8) {
            if (!int8_entropy_calibrator_cache_file.empty()) {
                if (exists(int8_entropy_calibrator_cache_file)) {
                    entropyCalibratorData = load_file(int8_entropy_calibrator_cache_file);
                    if (entropyCalibratorData.empty()) {
                        INFOE("entropyCalibratorFile is set as: %s, but we read is empty.", int8_entropy_calibrator_cache_file.c_str());
                        return false;
                    }
                    hasEntropyCalibrator = true;
                }
            }
            
            if (hasEntropyCalibrator) {
                if (!int8_images_folder.empty()) {
                    INFOW("int8_images_folder is ignore, when int8_entropy_calibrator_cache_file is set");
                }
            }
            else {
                entropyCalibratorFiles = glob_image_files(int8_images_folder);
                if (entropyCalibratorFiles.empty()) {
                    INFOE("Can not find any images(jpg/png/bmp/jpeg/tiff) from directory: %s", int8_images_folder.c_str());
                    return false;
                }

                if(entropyCalibratorFiles.size() < max_batch_size){
                    INFOW("Too few images provided, %d[provided] < %d[max batch size], image copy will be performed", entropyCalibratorFiles.size(), max_batch_size);
                    for(int i = entropyCalibratorFiles.size(); i < max_batch_size; ++i)
                        entropyCalibratorFiles.push_back(entropyCalibratorFiles[i % entropyCalibratorFiles.size()]);
                }
            }
        }
        else {
            if (hasEntropyCalibrator) {
                INFOW("int8_entropy_calibrator_cache_file is ignore, when Mode is '%s'", mode_string(mode));
            }
        }

        INFO("Compile %s %s.", mode_string(mode), source_onnx.c_str());
        shared_ptr<IBuilder> builder(createInferBuilder(gLogger), destroy_nvidia_pointer<IBuilder>);
        if (builder == nullptr) {
            INFOE("Can not create builder.");
            return false;
        }

        shared_ptr<IBuilderConfig> config(builder->createBuilderConfig(), destroy_nvidia_pointer<IBuilderConfig>);
        if (mode == Mode::FP16) {
            if (!builder->platformHasFastFp16()) {
                INFOW("Platform not have fast fp16 support");
            }
            config->setFlag(BuilderFlag::kFP16);
        }
        else if (mode == Mode::INT8) {
            if (!builder->platformHasFastInt8()) {
                INFOW("Platform not have fast int8 support");
            }
            config->setFlag(BuilderFlag::kINT8);
        }

        shared_ptr<INetworkDefinition> network;
        shared_ptr<nvonnxparser::IParser> onnxParser;
        const auto explicitBatch = 1U << static_cast<uint32_t>(nvinfer1::NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
        network = shared_ptr<INetworkDefinition>(builder->createNetworkV2(explicitBatch), destroy_nvidia_pointer<INetworkDefinition>);

        //from onnx is not markOutput
        onnxParser.reset(nvonnxparser::createParser(*network, gLogger), destroy_nvidia_pointer<nvonnxparser::IParser>);
        if (onnxParser == nullptr) {
            INFOE("Can not create parser.");
            return false;
        }

        if (!onnxParser->parseFromFile(source_onnx.c_str(), 1)) {
            INFOE("Can not parse OnnX file: %s", source_onnx.c_str());
            return false;
        }
        
        auto inputTensor = network->getInput(0);
        auto inputDims = inputTensor->getDimensions();

        shared_ptr<Int8EntropyCalibrator> int8Calibrator;
        if (mode == Mode::INT8) {
            auto calibratorDims = inputDims;
            calibratorDims.d[0] = max_batch_size;

            if (hasEntropyCalibrator) {
                INFO("Using exist entropy calibrator data[%d bytes]: %s", entropyCalibratorData.size(), int8_entropy_calibrator_cache_file.c_str());
                int8Calibrator.reset(new Int8EntropyCalibrator(
                    entropyCalibratorData, calibratorDims, int8process
                ));
            }
            else {
                INFO("Using image list[%d files]: %s", entropyCalibratorFiles.size(), int8_images_folder.c_str());
                int8Calibrator.reset(new Int8EntropyCalibrator(
                    entropyCalibratorFiles, calibratorDims, int8process
                ));
            }
            config->setInt8Calibrator(int8Calibrator.get());
        }

        INFO("Input shape is %s", join_dims(vector<int>(inputDims.d, inputDims.d + inputDims.nbDims)).c_str());
        INFO("Set max batch size = %d", max_batch_size);
        INFO("Set max workspace size = %.2f MB", max_workspace_size / 1024.0f / 1024.0f);

        int net_num_input = network->getNbInputs();
        INFO("Network has %d inputs:", net_num_input);
        vector<string> input_names(net_num_input);
        for(int i = 0; i < net_num_input; ++i){
            auto tensor = network->getInput(i);
            auto dims = tensor->getDimensions();
            auto dims_str = join_dims(vector<int>(dims.d, dims.d+dims.nbDims));
            INFO("      %d.[%s] shape is %s", i, tensor->getName(), dims_str.c_str());

            input_names[i] = tensor->getName();
        }

        int net_num_output = network->getNbOutputs();
        INFO("Network has %d outputs:", net_num_output);
        for(int i = 0; i < net_num_output; ++i){
            auto tensor = network->getOutput(i);
            auto dims = tensor->getDimensions();
            auto dims_str = join_dims(vector<int>(dims.d, dims.d+dims.nbDims));
            INFO("      %d.[%s] shape is %s", i, tensor->getName(), dims_str.c_str());
        }

        int net_num_layers = network->getNbLayers();
        INFO("Network has %d layers", net_num_layers);		
        builder->setMaxBatchSize(max_batch_size);
        config->setMaxWorkspaceSize(max_workspace_size);

        auto profile = builder->createOptimizationProfile();
        for(int i = 0; i < net_num_input; ++i){
            auto input = network->getInput(i);
            auto input_dims = input->getDimensions();
            input_dims.d[0] = 1;
            profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMIN, input_dims);
            profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kOPT, input_dims);
            input_dims.d[0] = max_batch_size;
            profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMAX, input_dims);
        }
        config->addOptimizationProfile(profile);

        INFO("Building engine...");
        auto time_start = chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count();
        shared_ptr<ICudaEngine> engine(builder->buildEngineWithConfig(*network, *config), destroy_nvidia_pointer<ICudaEngine>);
        if (engine == nullptr) {
            INFOE("engine is nullptr");
            return false;
        }

        if (mode == Mode::INT8) {
            if (!hasEntropyCalibrator) {
                if (!int8_entropy_calibrator_cache_file.empty()) {
                    INFO("Save calibrator to: %s", int8_entropy_calibrator_cache_file.c_str());
                    save_file(int8_entropy_calibrator_cache_file, int8Calibrator->getEntropyCalibratorData());
                }
                else {
                    INFO("No set entropyCalibratorFile, and entropyCalibrator will not save.");
                }
            }
        }

        auto time_end = chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count();
        INFO("Build done %lld ms !", time_end - time_start);
        
        // serialize the engine, then close everything down
        shared_ptr<IHostMemory> seridata(engine->serialize(), destroy_nvidia_pointer<IHostMemory>);
        return save_file(saveto, seridata->data(), seridata->size());
    }
};